#include "hip/hip_runtime.h"

#include "config.hpp"

#ifdef DIPOLAR_DIRECT_SUM

#include "cuda_utils.hpp"
#include <stdio.h>




//typedef float float;

__device__ inline void get_mi_vector_float(float res[3], float a[3], float b[3],float box_l[3],int periodic[3])
{
  int i;

  for(i=0;i<3;i++) {
    res[i] = a[i] - b[i];
#ifdef PARTIAL_PERIODIC
    if (periodic[i])
#endif
      res[i] -= floor(res[i]/box_l[i]+0.5)*box_l[i];
  }
}


//__device__ float scalar(float a[3], float b[3])
//{
// float sum=0.;
// for (int i=0;i<3;i++)
//  sum+=a[i]*b[i];
// return sum;
//}

#define scalar(a,b) (a[0]*b[0]+a[1]*b[1]+a[2]*b[2])



__device__ float dipole_ia(int id,float pf, float* r1, float *r2, float* dip1, float* dip2, float* f1, float* torque1, float* torque2, int force_flag, float box_l[3], int periodic[3])
{
//float dip1[3],dip2[3],r1[3],r2[3];
//for (int i=0;i<3;i++)
//{
// dip1[i]=_dip1[i];
// dip2[i]=_dip2[i];
// r1[i]=_r1[i];
// r2[i]=_r2[i];
//
//}
  float u,r_inv,pe1,pe2,pe3,pe4,r_sq,r3_inv,r5_inv,r_sq_inv,r7_inv,a,b,cc,d,ab;
#ifdef ROTATION
  float bx,by,bz,ax,ay,az; 
#endif
  float dr[3];
 
	
  // Distance between particles
  get_mi_vector_float(dr,r1,r2,box_l,periodic);

  // Powers of distance
  r_sq=scalar(dr,dr);
  r_sq_inv=1/r_sq;
//  if (id==248)
//  {
    //printf("xxx %g %g %g\n",dr[0],dr[1],dr[2]);
//  }
  r_inv=rsqrtf(r_sq);
  r3_inv=1/r_sq*r_inv;
  r5_inv=r3_inv*r_sq_inv;
  r7_inv=r5_inv*r_sq_inv;
 
  // Dot products
  pe1=scalar(dip1,dip2);
  pe2=scalar(dip1,dr);
  pe3=scalar(dip2,dr);
  pe4=3.0f*r5_inv;

  // Energy, if requested
  u= pf* ( pe1*r3_inv -   pe4*pe2*pe3);

  // Force, if requested
  if(force_flag) { 
    a=pe4*pe1;
    b=-15.0f*pe2*pe3*r7_inv;
    ab =a+b;
    cc=pe4*pe3;
    d=pe4*pe2;
    
    //  Result
    f1[0]=(pf*(ab*dr[0]+cc*dip1[0]+d*dip2[0]));
    f1[1]=(pf*(ab*dr[1]+cc*dip1[1]+d*dip2[1]));
    f1[2]=(pf*(ab*dr[2]+cc*dip1[2]+d*dip2[2]));
    
// Torques
#ifdef ROTATION
    ax=dip1[1]*dip2[2]-dip2[1]*dip1[2];
    ay=dip2[0]*dip1[2]-dip1[0]*dip2[2];
    az=dip1[0]*dip2[1]-dip2[0]*dip1[1];
    
    bx=dip1[1]*dr[2]-dr[1]*dip1[2];
    by=dr[0]*dip1[2]-dip1[0]*dr[2];
    bz=dip1[0]*dr[1]-dr[0]*dip1[1];
    
    torque1[0]=(pf*(-ax*r3_inv+bx*cc));
    torque1[1]=(pf *(-ay*r3_inv+by*cc));
    torque1[2]=(pf *(-az*r3_inv+bz*cc));
    
    
    bx=dip2[1]*dr[2]-dr[1]*dip2[2];
    by=dr[0]*dip2[2]-dip2[0]*dr[2];
    bz=dip2[0]*dr[1]-dr[0]*dip2[1];
	     
    torque2[0] =pf * (ax*r3_inv+bx*d);
    torque2[1] =pf * (ay*r3_inv+by*d);
    torque2[2] =pf * (az*r3_inv+bz*d);
    
#endif
  }    
	
  // Return energy
//  return u;
}


const int tileSize =16;






__global__ void DipolarDirectSum_kernel(float pf,
				     int n, float *pos, float* dip, float *f, float* torque, float box_l[3], int periodic[3]) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int step;


  if(i >= n)
    return;

  // Kahan summation based on the wikipedia article
  // Force
  float fi[3],fsum[3];
  
  // Torque
  float ti[3],tsum[3];

  float fj[3];
  float tj[3];



  // There is one thread per particle. Each thread computes interactions
  // with particles whose id is smaller than the thread id.
  // The force and torque of all the interaction partners of the current thread
  // is atomically added to global results ad once.
  // The result for the particle id equal to the thread id is atomically added
  // to global memory at the end.

  
  
  // Clear summation vars
  for (int j=0;j<3;j++)
  {
   // Force
   
   fsum[j]=0;
   // Torque
//   tc[j]=0;
   tsum[j]=0;
  }

// Loop

  

  for (int j=i+1;j<n;j++)
  {
      dipole_ia(i,1,pos+3*i,pos+3*j,dip+3*i,dip+3*j,fi,ti,tj,1,box_l,periodic);
//      printf("%d %d: %f %f %f\n",i,j,fi[0],fi[1],fi[2]); 
      for (int k=0;k<3;k++)
      {
        // Add rhs to global memory
//	printf("%d: Adding %f to %f \n",3*j+k,-fi[k], *(f+3*j+k));
        atomicAdd(f+3*j+k,-fi[k]);
//	printf("%d: now %f \n",3*j+k, *(f+3*j+k));

        atomicAdd((torque+3*j+k),tj[k]);
	tsum[k]+=ti[k];
	fsum[k]+=fi[k];
   } 

    
   
 }

 // Add the left hand side result to global memory
 for (int j=0;j<3;j++)
 {
  atomicAdd(f+3*i+j,fsum[j]);
  atomicAdd(torque+3*i+j,tsum[j]);
 }

 
}


void DipolarDirectSum_kernel_wrapper(float k, int n, float *pos, float *dip, float* f, float* torque, float box_l[3],int periodic[3]) {

  const int bs=64;
  dim3 grid(1,1,1);
  dim3 block(1,1,1);

  if(n == 0)
    return;

  if(n <= bs) {
    grid.x = 1;
    block.x = n;
  } else {
    grid.x = n/bs + 1;
    block.x = bs;
  }

  float* box_l_gpu;
  int* periodic_gpu;
  cuda_safe_mem(hipMalloc((void**)&box_l_gpu,3*sizeof(float)));
  cuda_safe_mem(hipMalloc((void**)&periodic_gpu,3*sizeof(int)));
  cuda_safe_mem(hipMemcpy(box_l_gpu,box_l,3*sizeof(float),hipMemcpyHostToDevice));
  cuda_safe_mem(hipMemcpy(periodic_gpu,periodic,3*sizeof(int),hipMemcpyHostToDevice));



  //printf("box_l: %f %f %f\n",box_l[0],box_l[1],box_l[2]);
  KERNELCALL(DipolarDirectSum_kernel,grid,block,(k, n, pos, dip,f,torque,box_l_gpu, periodic_gpu));
  hipFree(box_l_gpu);
  hipFree(periodic_gpu);

}




#endif
