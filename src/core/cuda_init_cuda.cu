/*
 * Copyright (C) 2010-2019 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>

#include "cuda_init.hpp"
#include "cuda_utils.cuh"

#include <utils/constants.hpp>

#include <cstring>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

#ifdef CUDA

/** \name minimally required compute capability. */
/**@{*/
static const int computeCapabilityMinMajor = 3;
static const int computeCapabilityMinMinor = 0;
/**@}*/

void cuda_init() {
  hipError_t error = hipStreamCreate(&stream[0]);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
}

/// get the number of CUDA devices.
int cuda_get_n_gpus() {
  int deviceCount;
  hipError_t error = hipGetDeviceCount(&deviceCount);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
  return deviceCount;
}

int cuda_check_gpu_compute_capability(int dev) {
  hipDeviceProp_t deviceProp;
  hipError_t error = hipGetDeviceProperties(&deviceProp, dev);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
  if (deviceProp.major < computeCapabilityMinMajor ||
      (deviceProp.major == computeCapabilityMinMajor &&
       deviceProp.minor < computeCapabilityMinMinor)) {
    return ES_ERROR;
  }
  return ES_OK;
}

void cuda_get_gpu_name(int dev, char name[64]) {
  hipDeviceProp_t deviceProp;
  hipError_t error = hipGetDeviceProperties(&deviceProp, dev);
  if (error != hipSuccess) {
    std::strncpy(name, "no GPU", 63);
    throw cuda_runtime_error_impl(error);
  }
  std::strncpy(name, deviceProp.name, 63);
  name[63] = 0;
}

EspressoGpuDevice cuda_get_device_props(const int dev) {
  hipDeviceProp_t deviceProp;
  hipError_t error = hipGetDeviceProperties(&deviceProp, dev);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
  EspressoGpuDevice device{dev,
                           "",
                           "",
                           -1,
                           deviceProp.major,
                           deviceProp.minor,
                           deviceProp.totalGlobalMem,
                           deviceProp.multiProcessorCount};
  std::strncpy(device.name, deviceProp.name, 64);
  device.name[63] = '\0';
  return device;
}

void cuda_set_device(int dev) {
  hipError_t error = hipSetDevice(dev);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
  error = hipStreamDestroy(stream[0]);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
  error = hipStreamCreate(&stream[0]);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
}

int cuda_get_device() {
  int dev;
  hipError_t error = hipGetDevice(&dev);
  if (error != hipSuccess) {
    throw cuda_runtime_error_impl(error);
  }
  return dev;
}

int cuda_test_device_access() {
  int *d = nullptr;
  int h = 42;
  hipError_t err;

  err = hipMalloc((void **)&d, sizeof(int));
  if (err != hipSuccess) {
    throw cuda_runtime_error_impl(err);
  }
  err = hipMemcpy(d, &h, sizeof(int), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    hipFree(d);
    throw cuda_runtime_error_impl(err);
  }
  h = 0;
  err = hipMemcpy(&h, d, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(d);
  if (err != hipSuccess) {
    throw cuda_runtime_error_impl(err);
  }
  if (h != 42) {
    return ES_ERROR;
  }
  return ES_OK;
}

#endif /* defined(CUDA) */
