#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012,2013 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "cuda_common.h"
extern "C" {

#include "config.h"
#include "random.h"
#include "particle_data.h"

  static void mpi_get_particles_lb(LB_particle_gpu *host_result);
  static void mpi_get_particles_slave_lb();
  static void mpi_send_forces_lb(LB_particle_force_gpu *host_forces);
  static void mpi_send_forces_slave_lb();
  
  static int max_ran = 1000000;
  static GPU_global_part_vars global_part_vars = {0,0,0};
  static __device__ __constant__ GPU_global_part_vars global_part_vars_gpu;
  
  /** struct for particle force */
  static LB_particle_force_gpu *particle_force = NULL;
  /** struct for particle position and veloctiy */
  static LB_particle_gpu *particle_data = NULL;
  /** struct for storing particle rn seed */
  static LB_particle_seed_gpu *part = NULL;

  LB_particle_gpu *host_data = NULL;
  
  /**cuda streams for parallel computing on cpu and gpu */
  extern hipStream_t stream[1];

  extern hipError_t err;
  extern hipError_t _err;
  
}


__device__ unsigned int getThreadIndex() {

  return blockIdx.y * gridDim.x * blockDim.x +
         blockDim.x * blockIdx.x +
         threadIdx.x;
}

/** kernel for the initalisation of the particle force array
 * @param *particle_force	Pointer to local particle force (Output)
 * @param *part			Pointer to the particle rn seed storearray (Output)
*/
__global__ void init_particle_force(LB_particle_force_gpu *particle_force, LB_particle_seed_gpu *part){

  unsigned int part_index = getThreadIndex();

  if(part_index<global_part_vars_gpu.number_of_particles){
    particle_force[part_index].f[0] = 0.0f;
    particle_force[part_index].f[1] = 0.0f;
    particle_force[part_index].f[2] = 0.0f;

    part[part_index].seed = global_part_vars_gpu.seed + part_index;
  }

}


/** kernel for the initalisation of the partikel force array
 * @param *particle_force	pointer to local particle force (Input)
*/
__global__ void reset_particle_force(LB_particle_force_gpu *particle_force){
	
  unsigned int part_index = getThreadIndex();
	
  if(part_index<global_part_vars_gpu.number_of_particles){
    particle_force[part_index].f[0] = 0.0f;
    particle_force[part_index].f[1] = 0.0f;
    particle_force[part_index].f[2] = 0.0f;
  }			
}


extern "C" {

  void cuda_enable_particle_communication() {
    global_part_vars.number_of_particles = n_total_particles;
    //TODO
  }

  /**setup and call particle reallocation from the host
   * @param *lbpar_gpu	Pointer to parameters to setup the lb field
   * @param **host_data	Pointer to host information data
  */
  void gpu_init_particle_comm( LB_particle_gpu **host_data ) {
    
    //we only run the function if there are new particles which have been created since the last call of this function
    if ( global_part_vars.number_of_particles == n_total_particles ) {
      
      global_part_vars.seed = (unsigned int)i_random(max_ran);
      global_part_vars.number_of_particles = n_total_particles;
      global_part_vars.communication_enabled = 1;

      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(global_part_vars_gpu), &global_part_vars, sizeof(GPU_global_part_vars)));

      if ( host_data )      hipHostFree(*host_data);
      if ( particle_force ) hipFree(particle_force);
      if ( particle_data )  hipFree(particle_data);
      if ( part )           hipFree(part);

    #if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200
      /**pinned memory mode - use special function to get OS-pinned memory*/
      hipHostAlloc((void**)host_data, global_part_vars.number_of_particles * sizeof(LB_particle_gpu), hipHostMallocWriteCombined);
    #else
      hipHostMalloc((void**)host_data, global_part_vars.number_of_particles * sizeof(LB_particle_gpu));
    #endif

      cuda_safe_mem(hipMalloc((void**)&particle_force, global_part_vars.number_of_particles * sizeof(LB_particle_force_gpu)));
      cuda_safe_mem(hipMalloc((void**)&particle_data, global_part_vars.number_of_particles * sizeof(LB_particle_gpu)));
      cuda_safe_mem(hipMalloc((void**)&part, global_part_vars.number_of_particles * sizeof(LB_particle_seed_gpu)));
      
      /** values for the particle kernel */
      int threads_per_block_particles = 64;
      int blocks_per_grid_particles_y = 4;
      int blocks_per_grid_particles_x = (global_part_vars.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
      dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

      if ( global_part_vars.number_of_particles )
        KERNELCALL(init_particle_force, dim_grid_particles, threads_per_block_particles, (particle_force, part));

    }
  }

  void lb_get_particle_pointer(LB_particle_gpu** pointeradress) {
    *pointeradress = particle_data;
  }

  void lb_get_particle_force_pointer(LB_particle_force_gpu** pointeradress) {
    *pointeradress = particle_force;
  }

  void copy_part_data_to_gpu() {

    mpi_get_particles_lb(host_data);

    /** get espresso md particle values*/
    hipMemcpyAsync(particle_data, host_data, global_part_vars.number_of_particles * sizeof(LB_particle_gpu), hipMemcpyHostToDevice, stream[0]);
  }


  /** setup and call kernel to copy particle forces to host
   * @param *host_forces contains the particle force computed on the GPU
  */
  void lb_copy_forces_GPU(LB_particle_force_gpu *host_forces){

    /** Copy result from device memory to host memory*/
    hipMemcpy(host_forces, particle_force, global_part_vars.number_of_particles * sizeof(LB_particle_force_gpu), hipMemcpyDeviceToHost);

      /** values for the particle kernel */
    int threads_per_block_particles = 64;
    int blocks_per_grid_particles_y = 4;
    int blocks_per_grid_particles_x = (global_part_vars.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
    dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

    /** reset part forces with zero*/
    KERNELCALL(reset_particle_force, dim_grid_particles, threads_per_block_particles, (particle_force));
	
    hipDeviceSynchronize();
  }


  /*************** REQ_GETPARTS ************/
  static void mpi_get_particles_lb(LB_particle_gpu *host_data)
  {
    int n_part;
    int g, pnode;
    Cell *cell;
    int c;
    MPI_Status status;

    int i;	
    int *sizes;
    sizes = malloc(sizeof(int)*n_nodes);

    n_part = cells_get_n_particles();

    /* first collect number of particles on each node */
    MPI_Gather(&n_part, 1, MPI_INT, sizes, 1, MPI_INT, 0, comm_cart);

    /* just check if the number of particles is correct */
    if(this_node > 0){
      /* call slave functions to provide the slave datas */
      mpi_get_particles_slave_lb();
    }
    else {
      /* master: fetch particle informations into 'result' */
      g = 0;
      for (pnode = 0; pnode < n_nodes; pnode++) {
        if (sizes[pnode] > 0) {
          if (pnode == 0) {
            for (c = 0; c < local_cells.n; c++) {
              Particle *part;
              int npart;	
              int dummy[3] = {0,0,0};
              double pos[3];
              cell = local_cells.cell[c];
              part = cell->part;
              npart = cell->n;
              for (i=0;i<npart;i++) {
                memcpy(pos, part[i].r.p, 3*sizeof(double));
                fold_position(pos, dummy);
                host_data[i+g].p[0] = (float)pos[0];
                host_data[i+g].p[1] = (float)pos[1];
                host_data[i+g].p[2] = (float)pos[2];
								
                host_data[i+g].v[0] = (float)part[i].m.v[0];
                host_data[i+g].v[1] = (float)part[i].m.v[1];
                host_data[i+g].v[2] = (float)part[i].m.v[2];
                
  #ifdef LB_ELECTROHYDRODYNAMICS
                host_data[i+g].mu_E[0] = (float)part[i].p.mu_E[0];
                host_data[i+g].mu_E[1] = (float)part[i].p.mu_E[1];
                host_data[i+g].mu_E[2] = (float)part[i].p.mu_E[2];
  #endif

  #ifdef ELECTROSTATICS
                if (coulomb.method == COULOMB_P3M_GPU) {
                  host_data[i+g].q = (float)part[i].p.q;
                }
  #endif
              }  
              g += npart;
            }  
          }
          else {
            MPI_Recv(&host_data[g], sizes[pnode]*sizeof(LB_particle_gpu), MPI_BYTE, pnode, REQ_GETPARTS,
            comm_cart, &status);
            g += sizes[pnode];
          }
        }
      }
    }
    COMM_TRACE(fprintf(stderr, "%d: finished get\n", this_node));
    free(sizes);
  }

  static void mpi_get_particles_slave_lb(){
   
    int n_part;
    int g;
    LB_particle_gpu *host_data_sl;
    Cell *cell;
    int c, i;

    n_part = cells_get_n_particles();

    COMM_TRACE(fprintf(stderr, "%d: get_particles_slave, %d particles\n", this_node, n_part));

    if (n_part > 0) {
      /* get (unsorted) particle informations as an array of type 'particle' */
      /* then get the particle information */
      host_data_sl = malloc(n_part*sizeof(LB_particle_gpu));
      
      g = 0;
      for (c = 0; c < local_cells.n; c++) {
        Particle *part;
        int npart;
        int dummy[3] = {0,0,0};
        double pos[3];
        cell = local_cells.cell[c];
        part = cell->part;
        npart = cell->n;

        for (i=0;i<npart;i++) {
          memcpy(pos, part[i].r.p, 3*sizeof(double));
          fold_position(pos, dummy);	
			
          host_data_sl[i+g].p[0] = (float)pos[0];
          host_data_sl[i+g].p[1] = (float)pos[1];
          host_data_sl[i+g].p[2] = (float)pos[2];

          host_data_sl[i+g].v[0] = (float)part[i].m.v[0];
          host_data_sl[i+g].v[1] = (float)part[i].m.v[1];
          host_data_sl[i+g].v[2] = (float)part[i].m.v[2];
          
  #ifdef LB_ELECTROHYDRODYNAMICS
          host_data_sl[i+g].mu_E[0] = (float)part[i].p.mu_E[0];
          host_data_sl[i+g].mu_E[1] = (float)part[i].p.mu_E[1];
          host_data_sl[i+g].mu_E[2] = (float)part[i].p.mu_E[2];
  #endif

  #ifdef ELECTROSTATICS
          if (coulomb.method == COULOMB_P3M_GPU) {
            host_data_sl[i+g].q = (float)part[i].p.q;
          }
  #endif
        }
        g+=npart;
      }
      /* and send it back to the master node */
      MPI_Send(host_data_sl, n_part*sizeof(LB_particle_gpu), MPI_BYTE, 0, REQ_GETPARTS, comm_cart);
      free(host_data_sl);
    }  
  }

  static void mpi_send_forces_lb(LB_particle_force_gpu *host_forces){
	
    int n_part;
    int g, pnode;
    Cell *cell;
    int c;
    int i;	
    int *sizes;
    sizes = malloc(sizeof(int)*n_nodes);
    n_part = cells_get_n_particles();
    /* first collect number of particles on each node */
    MPI_Gather(&n_part, 1, MPI_INT, sizes, 1, MPI_INT, 0, comm_cart);

    /* call slave functions to provide the slave datas */
    if(this_node > 0) {
      mpi_send_forces_slave_lb();
    }
    else{
    /* fetch particle informations into 'result' */
    g = 0;
      for (pnode = 0; pnode < n_nodes; pnode++) {
        if (sizes[pnode] > 0) {
          if (pnode == 0) {
            for (c = 0; c < local_cells.n; c++) {
              int npart;	
              cell = local_cells.cell[c];
              npart = cell->n;
              for (i=0;i<npart;i++) {
                cell->part[i].f.f[0] += (double)host_forces[i+g].f[0];
                cell->part[i].f.f[1] += (double)host_forces[i+g].f[1];
                cell->part[i].f.f[2] += (double)host_forces[i+g].f[2];
              }
   	    g += npart;
            }
          }
          else {
          /* and send it back to the slave node */
          MPI_Send(&host_forces[g], sizes[pnode]*sizeof(LB_particle_force_gpu), MPI_BYTE, pnode, REQ_GETPARTS, comm_cart);			
          g += sizes[pnode];
          }
        }
      }
    }
    COMM_TRACE(fprintf(stderr, "%d: finished send\n", this_node));

    free(sizes);
  }

  static void mpi_send_forces_slave_lb(){

    int n_part;
    LB_particle_force_gpu *host_forces_sl;
    Cell *cell;
    int c, i;
    MPI_Status status;

    n_part = cells_get_n_particles();

    COMM_TRACE(fprintf(stderr, "%d: send_particles_slave, %d particles\n", this_node, n_part));


    if (n_part > 0) {
      int g = 0;
      /* get (unsorted) particle informations as an array of type 'particle' */
      /* then get the particle information */
      host_forces_sl = malloc(n_part*sizeof(LB_particle_force_gpu));
      MPI_Recv(host_forces_sl, n_part*sizeof(LB_particle_force_gpu), MPI_BYTE, 0, REQ_GETPARTS,
      comm_cart, &status);
      for (c = 0; c < local_cells.n; c++) {
        int npart;	
        cell = local_cells.cell[c];
        npart = cell->n;
        for (i=0;i<npart;i++) {
          cell->part[i].f.f[0] += (double)host_forces_sl[i+g].f[0];
          cell->part[i].f.f[1] += (double)host_forces_sl[i+g].f[1];
          cell->part[i].f.f[2] += (double)host_forces_sl[i+g].f[2];
        }
        g += npart;
      }
      free(host_forces_sl);
    } 
  }
  /*@}*/
  
}
