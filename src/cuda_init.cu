/*
  Copyright (C) 2010,2011 The ESPResSo project
  
  This file is part of ESPResSo.
  
  ESPResSo is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  
  ESPResSo is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/
#include <hip/hip_runtime.h>

// CUDA code is always interpreted as C++, so we need the extern C interface
extern "C" {

#include "utils.h"
#include "parser.h"
#include "cuda_init.h"

}

static int list_gpus(Tcl_Interp *interp)
{
  int deviceCount, dev;

  if (hipGetDeviceCount(&deviceCount) != hipSuccess) {
    Tcl_AppendResult(interp, "cannot initialize CUDA", NULL);
    return TCL_ERROR;
  }

  // look for devices with compute capability > 1.1 (for atomic operations)
  for (dev = 0; dev < deviceCount; ++dev) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    if (deviceProp.major > 1 || (deviceProp.major == 1 && deviceProp.minor >= 1)) {
      char id[4 + 64 + TCL_INTEGER_SPACE];
      sprintf(id, " {%d %.64s}", dev, deviceProp.name);
      Tcl_AppendResult(interp, id, NULL);
    }
  }
  return TCL_OK;
}

int tclcommand_cuda(ClientData data, Tcl_Interp *interp,
		    int argc, char **argv)
{
  if (argc <= 1) {
    Tcl_AppendResult(interp, "too few arguments to the cuda command", (char *)NULL);
    return TCL_ERROR;
  }
  argc--; argv++;
  
  if (ARG0_IS_S("list")) {
    if (argc != 1) {
      Tcl_AppendResult(interp, "cuda list takes no arguments", (char *)NULL);
      return TCL_ERROR;
    }
    return list_gpus(interp);
  }
  else if (ARG0_IS_S("setdevice")) {
    int dev;
    hipError_t error;
    if (argc <= 1 || !ARG1_IS_I(dev)) {
      Tcl_AppendResult(interp, "expected: cuda setdevice <devnr>", (char *)NULL);
      return TCL_ERROR;
    }
    error = hipSetDevice(dev);
    if (error == hipSuccess) {
      return TCL_OK;
    }
    else {
      Tcl_AppendResult(interp, hipGetErrorString(error), (char *)NULL);
      return TCL_ERROR;
    }
  }
  else if (ARG0_IS_S("getdevice")) {
    if (argc != 1) {
      Tcl_AppendResult(interp, "cuda getdevice takes no arguments", (char *)NULL);
      return TCL_ERROR;
    }
    int dev;
    hipError_t error;
    error = hipGetDevice(&dev);
    if (error == hipSuccess) {
      char buffer[TCL_INTEGER_SPACE];
      sprintf(buffer, "%d", dev);
      Tcl_AppendResult(interp, buffer, (char *)NULL);
      return TCL_OK;
    }
    else {
      Tcl_AppendResult(interp, hipGetErrorString(error), (char *)NULL);
      return TCL_ERROR;
    }
  }
  else {
    Tcl_AppendResult(interp, "unknown subcommand \"", argv[0], "\"", (char *)NULL);
    return TCL_ERROR;
  }
}
