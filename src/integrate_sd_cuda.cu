#include "hip/hip_runtime.h"
/*
   Copyright (C) 2010,2011,2012 The ESPResSo project

   This file is part of ESPResSo.

   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "config.hpp"
#ifdef CUDA /* Terminates at end of file */

/* This is where the hydro dynamic interaction is implemented */

// TODO:
// * use preconditioner in iterative solver
// * implement matrix-free farfield (via fft)
// * add brownian motion

typedef double real;

#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include "assert.h"
#include "integrate_sd_cuda_debug.cuh"
#include "integrate_sd.hpp" // this includes magma and cublas
#include "cuda_utils.hpp"
#include "errorhandling.hpp"
#include "global.hpp"

const int numThreadsPerBlock = 32;

void _cudaCheckError(const char *msg, const char * file, const int line);
#define cudaCheckError(msg)  _cudaCheckError((msg),__FILE__,__LINE__)

#define myindex(i,j) ((i)*(lda)+(j))

#define cublasCall(call) stat=(call);assert(stat==HIPBLAS_STATUS_SUCCESS)

#define SD_RESISTANCE_CORRECT

/* ************************************* *
 * *******   private functions   ******* *
 * ************************************* */
void sd_compute_displacement(hipblasHandle_t cublas, double * r_d, int N, double eta, double a, double * L_d, 
			     double * total_mobility_d, double * force_d, double * disp_d);
//void sd_compute_mobility(hipblasHandle_t cublas, double * r_d, int N, double eta, double a, double * L_d, double * total_mobility_d);


// this solves iteratively using CG
// disp * (1+resistance*mobility) = mobility_d *  force_d 
// and returnes disp
// mobility and resistance are square matrizes with size <size> and lda <((size+31)/32)*32>
// force and disp are vectors of size <size>
void sd_iterative_solver(hipblasHandle_t cublas, const double * mobility, const double * resistance, const double * force, int size, double * disp);

// This computes the farfield contribution.
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// mobility is the mobility matrix which will be retruned
// L is the boxlength
__global__ void sd_compute_mobility_matrix(double * r, int N, double self_mobility, double a, double * L, double * mobility);

// adds to each of the diagonal elemnts of the sizse*size matrix matrix
// with lda lda 1
__global__ void sd_add_identity_matrix(double * matrix, int size, int lda);
void _cudaCheckError(const char *msg, const char * file, const int line);
// this computes the near field
// it calculates the ResistanceMatrix
__global__ void sd_compute_resistance_matrix(double * r, int N, double self_mobility, double a, double * L, double * resistance);

// make sure to have one thread per particle
__global__ void sd_real_integrate_prepare( double * r_d , double * disp_d, double * L, double a, int N);
__global__ void sd_real_integrate( double * r_d , double * disp_d, double * L, double a, int N);


// this sets a block to zero
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
__global__ void sd_set_zero_matrix(double * matrix, int size);


// this sets a block to zero
// data  : pointer to the given data
// size  : the size of the data
__global__ void sd_set_zero(double * data, int size);

// implementation of a bucket sort algorithm
// puts all the N particles with given position pos 
// and particle radius a within the periodic boundary 
// conditions of boxSize L_i = bucketSize_i * bucketNum_i
// puts them in the list particleList
// pos                device array of particle position xyz
// bucketSize         device array with the number of buckets in x y and z direction
// bucketNum          device array with the size of a bucket in x y and z direction
// N                  number of particles
// particleCount      device array of the numbers of particles per bucket. must be initalized to zero
// particleList       device array of the partilces in each bucket
// maxParticlePerCell maximum particles per cell
// totalBucketNUm     bucketNum[0]*bucketNum[1]*bucketNum[2] - the total number of buckets
__global__ void sd_bucket_sort( double * pos , double * bucketSize, int * bucketNum, int N,
				int * particleCount, int * particleList, int maxParticlePerCell, int totalBucketNum);

// BICGSTAB-Solver
// implimented as given in Numerik linearer Gleichungssysteme by Prof. Dr. Andreas Meister
// this solves A*x=b
// cublas a handle for cublas
// size   the size n of the matrix
// A      the given n*n matrix (in)
// lda    the leading demension of A
// b      the given solution vector (in)
// tol    requested tolerance of the solution
// maxit  maximum number of iterations
// x      the requested solution with an initial guess (in/out)
// returns 0 on success, else error code
int sd_bicgstab_solver(hipblasHandle_t cublas ,int size, real * A,int lda, real * b, real tol, int maxit, real * x);
/* *************************************************************************************************************** *
 * ********************************************     implementation    ******************************************** *
 * *************************************************************************************************************** */
/* *************************************************************************************************************** *
 * *******     III MM   MM PPP  L     EEEE MM   MM EEEE NN    N TTTTTTT  AAA  TTTTTTT III  OOO  NN    N    ******* *
 * *******      I  M M M M P  P L     E    M M M M E    N N   N    T    A   A    T     I  O   O N N   N    ******* *
 * *******      I  M  M  M PPP  L     EEE  M  M  M EEE  N  N  N    T    AAAAA    T     I  O   O N  N  N    ******* *
 * *******      I  M     M P    L     E    M     M E    N   N N    T    A   A    T     I  O   O N   N N    ******* *
 * *******     III M     M P    LLLL  EEEE M     M EEEE N    NN    T    A   A    T    III  OOO  N    NN    ******* *
 * *************************************************************************************************************** */
/* *************************************************************************************************************** */


// this calls all the functions to:
//  * generate the mobility matrix (farfield and nearfield)
//  * compute the displacements
//  * add the displacements to the positions
// TODO: add brownian motion, which is currently missing
// PARAMTERS:
// box_l_h : the size of the box in x,y and z-direction, on the host (in)
// N       : Number of particles (in)
// pos_h   : position of the particles, simple* array on host (in and out)
// force_h : forces on the particles, simple* array on host (in)
// velo_h  : velocities of the particles, simple* array on host (in and out)
// * : a simple array is e.g. [x_1, y_1, z_1, x_2, y_2, z_2, ...]
void propagate_pos_sd_cuda(double * box_l_h, int N,double * pos_h, double * force_h, double * velo_h){
  //printVectorHost(pos_h,3*N,"pos after call");
  double viscosity=sd_viscosity;
  double radius   =sd_radius;
  if (viscosity  < 0){
    std::cerr << "The viscosity for SD was not set\n";
    errexit();
  }
  if (radius  < 0){
    std::cerr << "The particle radius for SD was not set\n";
    errexit();
  }
  if (time_step < 0){
    std::cerr << "The timestep was not set\n";
    errexit();
  }
  
  int lda=((3*N+31)/32)*32;
  
  static hipblasHandle_t cublas=NULL;
  if (cublas==NULL){
    if (hipblasCreate(&cublas) != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "CUBLAS initialization failed\n";
      errexit();
    }
    //magma_init();
  }

  double * box_l_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&box_l_d, 3*sizeof(double)));
  cuda_safe_mem(hipMemcpy(box_l_d,box_l_h,3*sizeof(double),hipMemcpyHostToDevice));
  double * pos_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&pos_d, (DIM)*N*sizeof(double)));
  cuda_safe_mem(hipMemcpy(pos_d,pos_h,N*DIM*sizeof(double),hipMemcpyHostToDevice));
  //printVectorDev(pos_d,3*N,"pos after copy");
  double * force_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&force_d, DIM*N*sizeof(double)));
  cuda_safe_mem(hipMemcpy(force_d,force_h,N*DIM*sizeof(double),hipMemcpyHostToDevice));
  double * mobility_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&mobility_d, lda*N*3*sizeof(double)));
  double * disp_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&disp_d, DIM*N*sizeof(double)));
  cuda_safe_mem(hipMemcpy(disp_d,velo_h,N*DIM*sizeof(double),hipMemcpyHostToDevice));
  // rescale forces - this should not be done somewhere else ...
  hipblasStatus_t stat;
  double alpha=time_step;
  cublasCall(hipblasDscal( cublas, 3*N, &alpha, force_d, 1));
  //alpha=1/time_step;
  //cublasCall(hipblasDscal(cublas, DIM*N, &alpha, disp_d, 1));  
  sd_compute_displacement(cublas, pos_d, N, viscosity, radius, box_l_d, mobility_d, force_d, disp_d);
     
  //int numThreadsPerBlock = 3;
  int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;
  //stat = hipblasDaxpy(cublas, DIM*N, &alpha, v_d, 1, xr_d, 1);
  //assert(stat==HIPBLAS_STATUS_SUCCESS);
  alpha=time_step;
  cublasCall(hipblasDscal( cublas, 3*N, &alpha, disp_d, 1));  
  sd_real_integrate_prepare<<< numBlocks , numThreadsPerBlock  >>>(pos_d , disp_d, box_l_d, sd_radius, N);
  sd_real_integrate<<< numBlocks , numThreadsPerBlock  >>>(pos_d , disp_d, box_l_d, sd_radius, N);
  
  // copy back the positions
  cuda_safe_mem(hipMemcpy(pos_h,pos_d,N*DIM*sizeof(double),hipMemcpyDeviceToHost));
  // save the displacements as velocities (maybe somebody is interested)
  alpha=1/time_step;
  cublasCall(hipblasDscal(cublas, DIM*N, &alpha, disp_d, 1));
  cuda_safe_mem(hipMemcpy(velo_h,disp_d,N*DIM*sizeof(double),hipMemcpyDeviceToHost));
  
  
  cuda_safe_mem(hipFree((void*)box_l_d));
  cuda_safe_mem(hipFree((void*)pos_d));
  cuda_safe_mem(hipFree((void*)force_d));
  cuda_safe_mem(hipFree((void*)mobility_d));
  cuda_safe_mem(hipFree((void*)disp_d));
}



// calculate the farfield and the nearfield and add them
// PARAMETERS:
// cublas : a valid handle of cublas (in)
// r_d    : position of the particles on the device, size 3*N (in)
//          the form has to be [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N      : Number of particles (in)
// eta    : viscositiy of the fluid (in)
// a      : Particle radius (in)
// L_d    : boxsize in x y and z-directions (in)
// total_mobility_d: matrix of the computed total mobility, size 3*3*N*N (in/out, is overwritten)
void sd_compute_displacement(hipblasHandle_t cublas, double * r_d, int N, double eta, double a, double * L_d, 
			     double * total_mobility_d, double * force_d, double * disp_d)
{
  hipDeviceSynchronize(); // just for debugging
  cudaCheckError("");
  int lda=((3*N+31)/32)*32;
  //int numThreadsPerBlock = 32;
  int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;
  
  // compute the mobility Matrix
  double * helper_d=NULL;
  int ressize=max(4000,lda*DIM*N); // ressize has to be larger for small matrizes (else magma complains)
  cuda_safe_mem(hipMalloc( (void**)&helper_d, ressize*sizeof(double) ));
  assert(helper_d);
  double * mobility_d=NULL;
  cuda_safe_mem(hipMalloc( (void**)&mobility_d, lda*DIM*N*sizeof(double) ));
  assert(mobility_d);
  //printMatrixDev(mobility_d,3*N,3*N,"before mobility:");
  //printVectorDev(r_d,3*N,"positions");
  sd_set_zero_matrix<<<numBlocks, numThreadsPerBlock >>>(mobility_d,3*N);
  sd_compute_mobility_matrix<<< numBlocks , numThreadsPerBlock  >>>(r_d,N,1./(6.*M_PI*eta*a), a, L_d, mobility_d);
  hipDeviceSynchronize(); // just for debugging
  //printMatrixDev(mobility_d,3*N,3*N,"mobility_d");
  cudaCheckError("compute mobility error");
  //printMatrixDev(mobility_d,3*N,3*N,"early mobility:");
  // compute the resistance matrix
  double * resistance_d=NULL;
  cuda_safe_mem(hipMalloc( (void**)&resistance_d, ressize*sizeof(double) )); //this needs to be bigger for matrix inversion
  assert(resistance_d !=NULL);
  sd_set_zero_matrix<<<numBlocks, numThreadsPerBlock >>>(resistance_d,3*N);
  sd_compute_resistance_matrix<<< numBlocks , numThreadsPerBlock  >>>(r_d,N,1./(6.*M_PI*eta*a), a, L_d, resistance_d);
  hipDeviceSynchronize(); // we need both matrices to continue;
  cudaCheckError("compute resistance or mobility error");
  assert(!hasAnyNanDev(mobility_d,N*3*lda));
  assert(!hasAnyNanDev(resistance_d,N*3*lda));
  assert(isSymmetricDev(resistance_d,lda,N*3));
  //hipblasStatus_t status;
  
  //debug
  //printMatrixDev(mobility_d,lda,3*N,"mobility:");
  //printVectorDev(r_d,3*N,"position: ");
  //printMatrixDev(resistance_d,lda,3*N,"resitstance: ");

                                   
  sd_iterative_solver(cublas, mobility_d, resistance_d, force_d, 3*N,disp_d);
  
  

  /*double alpha=1, beta =0;
  status = hipblasDgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, DIM*N , DIM*N ,DIM*N, &alpha, mobility_d, lda,resistance_d, lda, &beta,helper_d, lda);
  assert(status == HIPBLAS_STATUS_SUCCESS);*/
  /*
  sd_add_identity_matrix<<< numBlocks , numThreadsPerBlock  >>>(helper_d,DIM*N,lda);
  hipDeviceSynchronize();
  
  cudaCheckError("add identity error");
  
  // inverting the matrix 
  int ipiv[DIM*N];
  int info;
  magma_dgetrf_gpu( DIM*N, DIM*N,helper_d, lda, ipiv, &info);
  assert(info==0);
  magma_dgetri_gpu( N*DIM, helper_d, lda, ipiv,resistance_d,ressize, &info);
  assert(info==0);
  // compute the inverse matrix
  // this is an alternative implementation ...
  // be sure to make sure everything else matches, because this one does not overwrite the original matrix
  // GPUGausSeidelDev(helper_d,inverse_d,DIM*N);
  hipDeviceSynchronize();
  cudaCheckError("inversion error");
  // compute total_mobility_d
  status = hipblasDgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, DIM*N , DIM*N ,DIM*N, &alpha, \
		       mobility_d, lda,helper_d, lda, &beta,total_mobility_d, lda);
		       assert(status == HIPBLAS_STATUS_SUCCESS);*/
  // free the two matrices again
  hipFree((void*)resistance_d);
  hipFree((void*)mobility_d);
  hipFree((void*)helper_d);
  cudaCheckError("in mobility");
}

// this calls magma functions to solve the problem: 
// disp * (1+resistance*mobility) = mobility_d *  force_d 
// and returnes disp
// mobility and resistance are square matrizes with size <size> and lda <((size+31)/32)*32>
// force and disp are vectors of size <size>
void sd_iterative_solver(hipblasHandle_t cublas, const double * mobility, const double * resistance, const double * force, int size, double * disp)
{
  int lda = ((size+31)/32)*32;
  double * mat_a = NULL;
  cuda_safe_mem(hipMalloc( (void**)&mat_a, lda*size*sizeof(double) ));       assert(mat_a != NULL);
  double * mat_a_bak = NULL;
  cuda_safe_mem(hipMalloc( (void**)&mat_a_bak, lda*size*sizeof(double) ));   assert(mat_a_bak != NULL);
  sd_set_zero_matrix<<<192,32>>>(mat_a,size);
  double * mob_force=NULL;
  cuda_safe_mem(hipMalloc( (void**)&mob_force, size*sizeof(double) ));       assert(mob_force !=NULL);
  double * result_checker=NULL;
  cuda_safe_mem(hipMalloc( (void**)&result_checker, size*sizeof(double) ));  assert(result_checker !=NULL);
  // vars for cuBLAS calls
  double alpha=1;
  double beta=0;
  // mat_a = (1+resistance*mobility)
  hipblasStatus_t stat;
  cublasCall(hipblasDgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, size , size ,size, &alpha, mobility, lda,resistance, lda, &beta,mat_a, lda));
  sd_add_identity_matrix<<<128,10>>>(mat_a,size,lda);// TODO: FIXME:  calculate something to set better values ...
  cuda_safe_mem(hipMemcpy(mat_a_bak, mat_a, lda*size*sizeof(double),hipMemcpyDeviceToDevice));
  // mob_force = mobility * force
  cublasCall(hipblasDgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mobility, lda, force, 1, &beta, mob_force, 1));
  int info;
  double res;
  //printVectorDev((double *)force,6,"Kraft");
  //printVectorDev(disp,6,"before");
  info = sd_bicgstab_solver(cublas ,size, mat_a,lda, mob_force, 1e-4, 10*size+100, disp);
  //printVectorDev(disp,6,"after");
  // compary to expected result
  cuda_safe_mem(hipMemcpy(mat_a, mat_a_bak, lda*size*sizeof(double),hipMemcpyDeviceToDevice));
  cublasCall(hipblasDgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, disp, 1, &beta, result_checker, 1));
  alpha=-1;
  cublasCall(hipblasDaxpy( cublas, size, &alpha, mob_force, 1, result_checker, 1));
  alpha=1;
  cublasCall(hipblasDdot( cublas, size, result_checker, 1, result_checker, 1,&res));
  if (info != 0){
    if (info == 1){
      fprintf(stderr, "Iterative solver did not fully converge ... the residuum was %6e\n\
We will continue anyway ...\n",res);
    }
    else{ // info == 2 || info == 4
      // try again with reseted displacement vector as initial guess
      sd_set_zero<<<192,16>>>(disp,size);
      info = sd_bicgstab_solver(cublas ,size, mat_a,lda, mob_force, 1e-4, 10*size+100, disp);
      //printVectorDev(disp,6,"after zeroing");
      if (info == 1){
	fprintf(stderr, "Iterative solver did not fully converge ... the residuum was %6e\n\
We will continue anyway ...\n",res);
      }
      else if (info == 2){
	fprintf(stderr, "Iterative solver failed ... the residuum was %6e\n\
We will continue but the results may be problematic ...\n",res);
      }
    }
    // dgetrs is not better - the contrary: results are worse ...
    /*int ipiv[size];
      magma_dgetrf_gpu( size, size,mat_a, lda, ipiv, &info);
      assert(info==0);
      magma_dgetrs_gpu('N', size, 1,
      mat_a, lda, ipiv,
      disp, size, &info);
      assert(info==0);
      // compary to expected result
      cuda_safe_mem(hipMemcpy(mat_a, mat_a_bak, lda*size*sizeof(double),hipMemcpyDeviceToDevice));
      cublasCall(hipblasDgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, disp, 1, &beta, result_checker, 1));
      alpha=-1;
      cublasCall(hipblasDaxpy( cublas, size, &alpha, mob_force, 1, result_checker, 1));
      alpha=1;
      cublasCall(hipblasDdot( cublas, size, result_checker, 1, result_checker, 1,&res));
      if (res > 1e-1){
      fprintf(stderr, "All methods failed :(. The residuum from getrs was %e\n",res);
      //cublasCall(hipblasDgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, disp, 1, &beta, result_checker, 1));
      //printVectorDev(mob_force, size, "mob_force");
      //printVectorDev(result_checker, size, "result_checker");
      //printVectorDev(disp, size, "disp");
      //printMatrixDev((double *)mobility,lda,size,"mobility");
      //printMatrixDev((double *)resistance,lda,size,"res");
      //printMatrixDev((double *)mat_a,lda,size,"mat_a");
      }*/
    //magma_int_t magma_dgetrs_gpu( magma_trans_t trans, magma_int_t n, magma_int_t nrhs,
    //				  double *dA, magma_int_t ldda, magma_int_t *ipiv,
    //				  double *dB, magma_int_t lddb, magma_int_t *info);
  }
  
  //assert(info==0);
  cuda_safe_mem(hipFree((void*)mat_a));
  cuda_safe_mem(hipFree((void*)mat_a_bak));
  cuda_safe_mem(hipFree((void*)mob_force));
  cuda_safe_mem(hipFree((void*)result_checker));
}
// this solves iteratively using CG
// disp * (1+resistance*mobility) = mobility_d *  force_d 
// and returnes disp
// mobility and resistance are square matrizes with size <size> and lda <((size+31)/32)*32>
// force and disp are vectors of size <size>
void sd_iterative_solver_own(hipblasHandle_t cublas, const double * mobility, const double * resistance, const double * force, int size, double * disp)
{
  int lda = ((size+31)/32)*32;
  double * mat_a = NULL;
  cuda_safe_mem(hipMalloc( (void**)&mat_a, lda*size*sizeof(double) ));  assert(mat_a != NULL);
  sd_set_zero_matrix<<<192,32>>>(mat_a,size);
  double * mob_force=NULL;
  cuda_safe_mem(hipMalloc( (void**)&mob_force, size*sizeof(double) ));  assert(mob_force !=NULL);
  double * resid=NULL;
  cuda_safe_mem(hipMalloc( (void**)&resid, size*sizeof(double) ));      assert(resid !=NULL);
  double * p=NULL;
  cuda_safe_mem(hipMalloc( (void**)&p, size*sizeof(double) ));          assert(p !=NULL);
  double * Ap=NULL;
  cuda_safe_mem(hipMalloc( (void**)&Ap, size*sizeof(double) ));         assert(Ap !=NULL);
  double rs_old;
  // count how many iterations we need
  int counter=0;
  assert(!hasAnyNanDev(mobility,size*lda));
  assert(!hasAnyNanDev(resistance,size*lda));
  // vars for cuBLAS calls
  double alpha=1;
  double beta=0;
  // mat_a = (1+resistance*mobility)
  hipblasStatus_t stat;
  cublasCall(hipblasDgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, size , size ,size, &alpha, mobility, lda,resistance, lda, &beta,mat_a, lda));
  sd_add_identity_matrix<<<128,10>>>(mat_a,size,lda);// TODO: FIXME:  calculate something to set better values ...
  // mob_force = mobility * force
  cublasCall(hipblasDgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mobility, lda, force, 1, &beta, mob_force, 1));
  //printMatrixDev(mat_a,lda,size,"A");
  // use mob_force as initial guess
  cublasCall(hipblasDcopy(cublas, size,mob_force,1,disp, 1));
  //resid = mob_force-mat_a * disp; //r = b-A*x
  alpha = -1;
  cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, disp, 1, &beta, resid, 1));
  printVectorDev(resid,size,"-A*disp");
  printVectorDev(mob_force,size,"solution");
  alpha = 1;
  cublasCall(hipblasDaxpy(cublas, size, &alpha, mob_force, 1, resid, 1));
  printVectorDev(resid,size,"residuum");
  
  // p = resid;                     //p=r
  cublasCall(hipblasDcopy(cublas, size,resid,1,p, 1));
  // rsquare_old = r * r;           //rsold=r*r
  cublasCall(hipblasDdot( cublas, size, resid, 1, resid, 1, &rs_old));
  std::cerr << counter <<" iterations in integrate_sd::inversion, residuum is "<<rs_old<<std::endl;
  const double req_prec=1e-4;
  if (sqrt(rs_old) < req_prec){
    printf("Converged immediatly\n");
    return;
  }
  while (true){
    // Ap = A * p
    beta = 0;  alpha = 1; cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, p, 1, &beta, Ap, 1));
    double pAp;
    cublasCall(hipblasDdot( cublas, size, p, 1, Ap, 1, &pAp));
    assert(!isnan(pAp));
    //                              //alpha=rsold/pAp
    alpha = rs_old / pAp;
    // disp += alpha * p            // x=x+alpha * p
    cublasCall(hipblasDaxpy(cublas, size, &alpha,  p, 1, disp, 1));
    // resid -= alpha * Ap;         // r=r-alpha * Ap
    double minusalpha=-alpha;
    cublasCall(hipblasDaxpy(cublas, size, &minusalpha, Ap, 1, resid, 1));
    double rs_new;
    // rs_new = r * r;              // rsnew = r*r
    cublasCall(hipblasDdot( cublas, size, resid, 1, resid, 1, &rs_new));
    if (sqrt(rs_new) < req_prec || counter > 2000){
      break;
    }
    // p=resid+rs_new/rs_old*p      // p = r+rsnew/rsold*p
    alpha = rs_new/rs_old;
    cublasCall(hipblasDscal( cublas, size, &alpha, p, 1));
    alpha=1;
    cublasCall(hipblasDaxpy( cublas, size, &alpha, resid, 1, p, 1));
    //                              // rsold=rsnew;
    rs_old=rs_new;
    counter++;
    if (counter % 100 == 0){
      std::cerr << counter <<" iterations in integrate_sd::inversion, residuum is "<<rs_new<<std::endl;
    }
  }
  printf("Converged after %d iterations\n",counter);
  cuda_safe_mem(hipFree((void*)mat_a));
  cuda_safe_mem(hipFree((void*)mob_force));
  cuda_safe_mem(hipFree((void*)resid));
  cuda_safe_mem(hipFree((void*)p));
  cuda_safe_mem(hipFree((void*)Ap));
}

// BICGSTAB-Solver
// implimented as given in Numerik linearer Gleichungssysteme by Prof. Dr. Andreas Meister
// this solves A*x=b
// cublas a handle for cublas
// size   the size n of the matrix
// A      the given n*n matrix (in)
// lda    the leading demension of A
// b      the given solution vector (in)
// tol    requested tolerance of the solution
// maxit  maximum number of iterations
// x      the requested solution with an initial guess (in/out)
// returns 0 on success, else error code
int sd_bicgstab_solver(hipblasHandle_t cublas ,int size, real * A,int lda, real * b, real tol, int maxit, real * x){
  // vector malloc
  real * r0=NULL;
  cuda_safe_mem(hipMalloc( (void**)&r0, size*sizeof(real) ));       assert(r0 != NULL);
  real * r=NULL;
  cuda_safe_mem(hipMalloc( (void**)&r, size*sizeof(real) ));        assert(r != NULL);
  real * p=NULL;
  cuda_safe_mem(hipMalloc( (void**)&p, size*sizeof(real) ));        assert(p != NULL);
  real * v=NULL;
  cuda_safe_mem(hipMalloc( (void**)&v, size*sizeof(real) ));        assert(v != NULL);
  real * t=NULL;
  cuda_safe_mem(hipMalloc( (void**)&t, size*sizeof(real) ));        assert(t != NULL);
  real * test=NULL;
  cuda_safe_mem(hipMalloc( (void**)&test, size*sizeof(real) ));     assert(test != NULL);
  // constants
  real eps;
  if (sizeof(real) == sizeof(double)){
    eps = 1e-15;
  } else {
    eps = 1e-7;
  }
  eps = min(eps,tol*1e-2);
  // other variables
  hipblasStatus_t stat;
  real alpha=1;
  real beta=0;
  real tolb;
  // compute the norm of b
  real normb;
  cublasCall(hipblasDdot( cublas, size, b, 1, b, 1, &normb));
  normb=sqrt(normb);
  //tolb=min(tol*size, tol*normb); // tol is not realy usefull as this wont be reached ... at least without preconditioning
  //tolb=max(normb*eps, tolb);
  tolb=tol*normb;
  // r0 = b-A*x
  alpha = -1;
  cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, x, 1, &beta, r0, 1));
  alpha = 1;
  cublasCall(hipblasDaxpy(cublas, size, &alpha, b, 1, r0, 1));
  // r = r0
  cublasCall(hipblasDcopy(cublas, size,r0,1,r, 1));
  // rr0 = r*r0
  real rr0;
  cublasCall(hipblasDdot( cublas, size, r0, 1, r0, 1, &rr0));
  // p =r
  cublasCall(hipblasDcopy(cublas, size,r0,1,p, 1));
  // normr=norm(r)
  real normr=sqrt(rr0);
  int iteration=0;
  real lastnorm=normr;
  real initnorm=normr;
  // check for conversion or max iterations
  while (iteration < maxit && normr >= tolb){
    // v=A*p
    cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, p, 1, &beta, v, 1));
    // vr0 = v*r0
    real vr0;
    cublasCall(hipblasDdot( cublas, size, v, 1, r0, 1, &vr0));
    if (fabs(vr0) < eps || rr0 == 0){
      if (fabs(vr0) < eps)
	fprintf(stderr, "BICGSTAB break-down.\n");
      else
	fprintf(stderr, "BICGSTAB solution stagnates.\n");
      cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
      if (tolb*100 > normr){
	return 1;
      } else {
	return 2;
      }
    }
    // alpha = rr0/vr0
    real myAlpha=rr0/vr0;
    real minusMyAlpha = -myAlpha;
    // s = r - alpha v
    //cublasCall(hipblasDcopy(cublas, size,r,1,s, 1));
    cublasCall(hipblasDaxpy(cublas, size, &minusMyAlpha, v, 1, r, 1)); //s->r
    // t = A * s
    cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, r, 1, &beta, t, 1));// s->r
    // ts = s * t
    real ts;
    cublasCall(hipblasDdot( cublas, size, t, 1, r, 1, &ts));// s->r
    // tt = t * t
    real tt;
    cublasCall(hipblasDdot( cublas, size, t, 1, t, 1, &tt));
    if (tt==0 || ts == 0){
      fprintf(stderr, "BICGSTAB break-down.\n");
      cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
      if (tolb*100 > normr){
	return 1;
      } else {
	return 2;
      }
    }
    // omega = ts/tt
    real myOmega=ts/tt;
    // x = x + alpha p + omega s
    cublasCall(hipblasDaxpy(cublas, size, &myAlpha, p, 1, x, 1));
    cublasCall(hipblasDaxpy(cublas, size, &myOmega, r, 1, x, 1));
    // copyback of s to r
    // r = s - omega t
    real minusMyOmega=-1*myOmega;
    cublasCall(hipblasDaxpy(cublas, size, &minusMyOmega, t, 1, r, 1));
    //myOmega*=-1;
    // r1r0 = r * r0
    real r1r0;
    cublasCall(hipblasDdot( cublas, size, r, 1, r0, 1, &r1r0));
    // beta = (alpha * r1r0 ) / (omega rr0)
    real myBeta = (myAlpha*r1r0)/(myOmega*rr0);
    // p = r + beta ( p - omega v)= beta p + r - beta omega v
    cublasCall(hipblasDscal(cublas, size, &myBeta, p, 1));
    cublasCall(hipblasDaxpy(cublas, size, &alpha, r, 1, p, 1));
    alpha=-myBeta*myOmega;
    cublasCall(hipblasDaxpy(cublas, size, &alpha, v, 1, p, 1));
    alpha=1;
    rr0=r1r0;
    real r1r1;
    cublasCall(hipblasDdot( cublas, size, r, 1, r, 1, &r1r1));
    normr=sqrt(r1r1);
    iteration++;
    if (lastnorm*sqrt(eps) > normr){ // restart
      //fprintf(stderr, "recalculation r\n");
      cublasCall(hipblasDcopy(cublas, size,b,1,r, 1));
      alpha=-1;beta=1;
      cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, x, 1, &beta, r, 1));
      alpha= 1;beta=0;
      cublasCall(hipblasDdot( cublas, size, r, 1, r, 1, &rr0));
      normr=sqrt(rr0);
      lastnorm = normr;
      // r = r0
      cublasCall(hipblasDcopy(cublas, size,r,1,r0, 1));
      // p =r
      cublasCall(hipblasDcopy(cublas, size,r,1,p, 1));
    }
    if (iteration%50000 == 0){ // enable debugging by setting this to a lower value
      real realnorm;
      {// recalculate normr
	cublasCall(hipblasDcopy(cublas, size,b,1,test, 1));
	alpha=-1;beta=1;
	cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, x, 1, &beta, test, 1));
	alpha= 1;beta=0;
	cublasCall(hipblasDdot( cublas, size, test, 1, test, 1, &realnorm));
	realnorm=sqrt(realnorm);
      }
      fprintf(stderr,"  Iteration: %6d Residuum: %12f RealResiduum: %12f\n",iteration, normr, realnorm);
    }
    if (initnorm*1e10 < normr){ // somehow our solution explodes ...
      fprintf(stderr, "BICGSTAB did not converge. Aborting.\n");
      cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
      return 4;
    }
  }
  // this should not be needed, as we restart ...
  /*{// recalculate normr
    cublasCall(hipblasDcopy(cublas, size,b,1,r, 1));
    alpha=-1;beta=1;
    cublasCall(hipblasDgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, x, 1, &beta, r, 1));
    alpha= 1;beta=0;
    real r1r1;
    cublasCall(hipblasDdot( cublas, size, r, 1, r, 1, &r1r1));
    normr=sqrt(r1r1);
    }*/
  if (normr > tolb*1.01){
    fprintf(stderr, "BICGSTAB solution did not converge after %d iterations. Error was %e1 %% to high.\n",iteration,(normr/tolb-1)*100);
    cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
    if (tolb*100 > normr){
      fprintf(stderr, "1: tolb: %e normr: %e \n",tolb, normr);
      return 1;
    } else {
      fprintf(stderr, "2: tolb: %e normr: %e \n",tolb, normr);
      return 2;
    }
  }
  //fprintf(stderr, "BICGSTAB solution did converge after %d iterations.\n",iteration);
  
  cuda_safe_mem(hipFree((void*)r0));
  cuda_safe_mem(hipFree((void*)r));
  cuda_safe_mem(hipFree((void*)p));
  cuda_safe_mem(hipFree((void*)v));
  cuda_safe_mem(hipFree((void*)t));
  cuda_safe_mem(hipFree((void*)test));
  return 0;
}


// This computes the farfield contribution of the mobility
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// mobility is the mobility matrix which will be retruned
// L_d is the boxlength
__global__ void sd_compute_mobility_matrix(double * r, int N, double self_mobility, double a, double * L_g, double * mobility){
  double mypos[3];
  const int lda=((3*N+31)/32)*32;
  __shared__ double L[3];
  __shared__ double cachedPos[3*numThreadsPerBlock];
  __shared__ double writeCache[3*numThreadsPerBlock];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < 3){ // copy L to shared memory
    L[threadIdx.x]=L_g[threadIdx.x];
  }
  __syncthreads();
  // get data for myposition - using coalscaled memory access
  for (int l=0;l<3;l++){
    cachedPos[numThreadsPerBlock*l+threadIdx.x] = r[numThreadsPerBlock*(l+blockIdx.x*3)+threadIdx.x];
  }
  __syncthreads();
  for (int l=0;l<3;l++){
    mypos[l] = cachedPos[threadIdx.x*3+l];
  }

  /*if (i < N){
    // first write the self contribution
#pragma unroll
    for (int k=0; k < DIM; k++){
      //#pragma unroll
      //for (int l=0; l < DIM; l++){
      //mobility[myindex(DIM*i+k,DIM*i+l)]=0;
      //}
      mobility[myindex(DIM*i+k,DIM*i+k)]=self_mobility;
    }
    }*/
  for (int offset=0;offset<N;offset+=numThreadsPerBlock){
    // copy positions to shared memory
#pragma unroll
    for (int l=0;l<3;l++){
      cachedPos[numThreadsPerBlock*l+threadIdx.x] = r[offset*3+numThreadsPerBlock*l+threadIdx.x];
    }
    __syncthreads();
    if (i < N){
      for (int j=offset;j<min(offset+numThreadsPerBlock,N);j++){
	// this destroys coascaled memory access ...
	/*if (i==j){
	  j++; //just continue with next particle
	  if (j==offset+numThreadsPerBlock){
	    continue;
	  }
	}*/
	/*if (i==j){
#pragma unroll 3
	  for (int l=0;l<3;l++){
	    writeCache[threadIdx.x*3+l]=0;
	  }
	}*/
	//if (j < N ){
	double dr[DIM];
	double dr2=0;
#pragma unroll 3
	for (int k=0;k<DIM;k++){
	  //dr[k]=r[DIM*i+k]-r[DIM*j+k]; // r_ij
	  dr[k]=mypos[k]-cachedPos[DIM*(j-offset)+k]; // r_ij
	  /*if (isnan(dr[k])){
	    dr[k]=1337;
	    }*/
	  dr[k]-=rint(dr[k]/L[k])*L[k]; // fold back
	  dr2+=dr[k]*dr[k];
	}
	dr2=max(dr2,0.01);
	double drn= sqrt(dr2); // length of dr
	double b = a/drn;
      
	/*if (0.5 < b){  // drn < 2*a
	  /*double t=3./32./drn/a*self_mobility;
	  double t2=(1-9./32.*drn/a)*self_mobility;
	  for (k=0; k < DIM; k++){
	  for (l=0;l < DIM; l++){
	  mobility[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*t;
	  }
	  mobility[myindex(DIM*i+k,DIM*j+k)]+=t2;
	  }*/ // this should not happen ...
	// python implementation:
	//T=one*(1-9./32.*drn/a)+3./32.*dr*drt/drn/a;
	//}
	double t,t2;
	// this also catches the case i == j
	if (0.5 < b){  // drn < 2*a
	  t=0;
	  t2=0;
	  if (i==j){
	    t2=self_mobility;
	  }
	} else {
	  double b2=b*b;
	  t=(0.75-1.5*b2)*b/dr2*self_mobility;
	  t2=(0.75+0.5*b2)*b*self_mobility;
	}
	//mobility[threadIdx.x]=3+threadIdx.x;
	double tmp_el13;
#pragma unroll 3
	for (int k=0; k < DIM; k++){
	  if (k ==0){ // these ifs should be removed at compile time ... after unrolling
#pragma unroll 3
	    for (int l=0;l < 3; l++){
	      //mobility[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*t;
	      writeCache[3*threadIdx.x+l]=dr[k]*dr[l]*t;
	    }
	  }
	  else if(k==1){
	    tmp_el13 = writeCache[3*threadIdx.x+2];
	    writeCache[3*threadIdx.x+0]=writeCache[3*threadIdx.x+1];
#pragma unroll 2
	    for (int l=1;l < DIM; l++){
	      //mobility[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*t;
	      writeCache[3*threadIdx.x+l]=dr[k]*dr[l]*t;
	    }	
	  }
	  else{
	    writeCache[3*threadIdx.x+0]=tmp_el13;
	    writeCache[3*threadIdx.x+1]=writeCache[3*threadIdx.x+2];
	    writeCache[3*threadIdx.x+2]=dr[k]*dr[2]*t;
	  }
	  writeCache[3*threadIdx.x+k]+=t2;
	    
	  __syncthreads();
	  //int max = min(blockDim.x, N-(blockIdx.x*blockDim.x));
	  int max = min(blockDim.x,N-blockDim.x*blockIdx.x);
	  for (int l=0;l<3;l++){
	    //mobility[(DIM*j+k)*3*N+blockIdx.x*blockDim.x+threadIdx.x+blockDim.x*l]=writeCache[threadIdx.x+blockDim.x*l];
	    mobility[(DIM*j+k)*lda+blockIdx.x*blockDim.x*3+max*l+threadIdx.x]=writeCache[max*l+threadIdx.x];
	  }
	  //mobility[myindex(DIM*i+k,DIM*j+k)]+=t2;
	}
	// python implementation:
	// T=one*(0.75+0.5*b2)*b+(0.75-1.5*b2)*b*drt*dr/dr2;
	//} // if (j <N)
      } // for (j = ...
    } // if (i < N)
  }// for offset = ...
}


// this computes the near field
// it calculates the ResistanceMatrix
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// L_d is the boxlength
// resistance is the resistance matrix which will be retruned
__global__ void sd_compute_resistance_matrix(double * r, int N, double self_mobility, double a, double * L_g, double * resistance){
  //__shared__ double myPos[3*numThreadsPerBlock];
  double mypos[3];
  __shared__ double L[3];
  __shared__ double cachedPos[3*numThreadsPerBlock];
  const int lda=(((N*3)+31)/32)*32;
  //__shared__ double myresistance[6*numThreadsPerBlock];
  double myresistance[6];
  //__shared__ double otherresistance[6*numThreadsPerBlock];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < 3){ // copy L to shared memory
    L[threadIdx.x]=L_g[threadIdx.x];
  }
  //__syncthreads();
  // get data for myposition - but coalscaled
  /*for (int l=0;l<3;l++){
    myPos[threadIdx.x+l*numThreadsPerBlock] = r[threadIdx.x+l*numThreadsPerBlock+blockIdx.x*blockDim.x*3];
    }*/
  for (int l=0;l<3;l++){
    cachedPos[threadIdx.x+l*numThreadsPerBlock] = r[threadIdx.x+l*numThreadsPerBlock+blockIdx.x*blockDim.x*3];
  }
  __syncthreads();
  for (int d=0;d<3;d++){
    mypos[d] = cachedPos[threadIdx.x*3+d];
  }
  
  //for (int i = idx; i < N; i+=blockDim.x*gridDim.x){
  /*if (i < N){
#pragma unroll 3
    for (int k=0; k < DIM; k++){
#pragma unroll 3
      for (int l=0;l < DIM; l++){
	resistance[myindex(DIM*i+k,DIM*i+l)]=0; // we will add some terms on the diagonal, so set it to zero before
      }
    }
  }*/
  for (int offset=0;offset<N;offset+=numThreadsPerBlock){
    // copy positions to shared memory
#pragma unroll
    for (int l=0;l<3;l++){
      cachedPos[threadIdx.x+l*numThreadsPerBlock] = r[threadIdx.x+l*numThreadsPerBlock+offset*3];
    }
    __syncthreads();
    for (int j=offset;j<min(offset+numThreadsPerBlock,N);j++){
      double dr[DIM];
      double dr2=0;
#pragma unroll
      for (int k=0;k<DIM;k++){
	dr[k]=mypos[k]-cachedPos[3*(j-offset)+k]; // r_ij
	dr[k]-=L[k]*rint(dr[k]/L[k]); // fold back
	dr2+=dr[k]*dr[k];
      }
#ifdef SD_RESISTANCE_CORRECT
      double r2bcorr_diag_self     = 0;
      double r2bcorr_diag_mix      = 0;
      double r2bcorr_offdiag_self  = 0;
      double r2bcorr_offdiag_mix   = 0;
#else
      double offdiag_fac=0;
      double diag_fac=0;
#endif
      if (i >= N || i ==j || j >= N){
	;
      }
      else if (dr2 < 4*a*4*a && 2*a*2*a < dr2 ){// check whether 2*a < drn < 4*a
	// python code:
	// # Use only singular therms, namely to order O(s_ij^0)                                                                  
	// T=(1./4./s-1/4-9./40.*ls)*dr*drt/dr2
	// #           ^ this additonal constant is so that the mobility is smooth
	// # c.f. N.-Q. Nguyen and A. J. C. Ladd, PHYSICAL REVIEW E 66, 046708 (2002) equation (34)                               
	// T+=1./6.*ls*(-one+dr*drt/dr2)
	// R[3*i:3*i+3,3*j:3*j+3]=-T
	// R[3*i:3*i+3,3*i:3*i+3]+=T
	double drn= sqrt(dr2); // length of dr
	double s = drn/a-2;
	double ls = log(s);
	
#ifdef SD_RESISTANCE_CORRECT
	double const t_c=-0.125+9./40.*log(2.)+3./112.*2.*log(2.);
	double offdiag_fac =(-0.25/s+9./40.*ls+3./112.*s*ls-t_c)/dr2;
	double diag_fac    =(1./6.*ls);
#else
	double const t_c=-0.125+9./40.*log(2.)+3./112.*2.*log(2.);
	double const t2_c=2./6.*log(2.);
	offdiag_fac =(-0.25/s+9./40.*ls+3./112.*s*ls-t_c)/dr2/self_mobility;
	diag_fac    =(1./6.*ls-t2_c)/self_mobility;
#endif
#ifdef SD_RESISTANCE_CORRECT
	double dr4=dr2*dr2;
	double dr6=dr4*dr2;
	// constants for correction
	const double dr_c1 = 4;
	const double dr_c2 = 4*4;
	const double dr_c3 = 4*4*4;
	const double dr_c4 = 4*4*4*4;
	const double dr_c5 = 4*4*4*4*4;
	const double dr_c6 = 4*4*4*4*4*4;
	const double r2bcorr_diag_self_c    = (4.*dr_c6)/(4.*dr_c6-9.*dr_c4+12.*dr_c2-4.)         ;
	const double r2bcorr_diag_mix_c     = (9.*dr_c5-4.*dr_c3)/(4.*dr_c6-9.*dr_c4+12.*dr_c2-4.);
	const double r2bcorr_offdiag_self_c = 16.*dr_c2 /(16.*dr_c2-25)                            - 2./6.*log(2.);
	const double r2bcorr_offdiag_mix_c  = 20.*dr_c1 /(16.*dr_c2-25)                            - 2./6.*log(2.);
	// real computation
	r2bcorr_diag_self     = diag_fac    - 1./(1-9./4./dr2+3./dr4-1./dr6)                     + r2bcorr_diag_self_c;
	r2bcorr_diag_mix      = diag_fac    - (6.*dr4*drn-4.*dr2*drn)/(4.*dr6-9.*dr4+12.*dr2-4.) + r2bcorr_diag_mix_c;
	r2bcorr_offdiag_self  = offdiag_fac - 1./(1.-25./16./dr2)                                + r2bcorr_offdiag_self_c;
	r2bcorr_offdiag_mix   = offdiag_fac - 1./(16./20.*drn-25./20./drn)                       + r2bcorr_offdiag_mix_c;
	r2bcorr_diag_self    /= self_mobility;
	r2bcorr_diag_mix     /= self_mobility;
	r2bcorr_offdiag_self /= self_mobility;
	r2bcorr_offdiag_mix  /= self_mobility;
#endif
      }
      if (i < N){
#pragma unroll 3
	for (int k=0; k < DIM; k++){
#pragma unroll 3
	  for (int l=0;l < DIM; l++){
#ifdef SD_RESISTANCE_CORRECT
	    resistance[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*r2bcorr_offdiag_mix;
#else
	    resistance[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*offdiag_fac;
#endif
	
	    //resistance[myindex(DIM*i+k,DIM*i+l)]-=dr[k]*dr[l]*t;
	  }
#ifdef SD_RESISTANCE_CORRECT
	  myresistance[k]-=dr[k]*dr[k]*r2bcorr_offdiag_self;
	  resistance[myindex(DIM*i+k,DIM*j+k)]+=r2bcorr_diag_mix;
	  myresistance[k]-=r2bcorr_diag_self;
#else
	  myresistance[k]-=dr[k]*dr[k]*offdiag_fac;
	  resistance[myindex(DIM*i+k,DIM*j+k)]+=diag_fac;
	  myresistance[k]-=diag_fac;
#endif
	}
      }
#ifdef SD_RESISTANCE_CORRECT
      myresistance[3]-=r2bcorr_offdiag_self*dr[0]*dr[1];
      myresistance[4]-=r2bcorr_offdiag_self*dr[0]*dr[2];
      myresistance[5]-=r2bcorr_offdiag_self*dr[1]*dr[2];
#else
      myresistance[3]-=offdiag_fac*dr[0]*dr[1];
      myresistance[4]-=offdiag_fac*dr[0]*dr[2];
      myresistance[5]-=offdiag_fac*dr[1]*dr[2];
#endif
      // python implementation:
      //T=one*(1-9./32.*drn/a)+3./32.*dr*drt/drn/a;
    }
    /*else{ // set the block to zero
    // it might be faster to set everything in the beginning to zero ...
    // or use sparse matrices ...
#pragma unroll 3
	  for (int k=0; k < DIM; k++){
#pragma unroll 3
	    for (int l=0;l < DIM; l++){
	      resistance[myindex(DIM*i+k,DIM*j+l)]=0;
	    }
	    }  
	  }*/
    
  }
  if ( i < N){
#pragma unroll
    for (int k=0;k<3;k++){
      resistance[myindex(DIM*i+k,DIM*i+k)]=myresistance[k];
    }
    resistance[myindex(DIM*i+0,DIM*i+1)]=myresistance[3];
    resistance[myindex(DIM*i+1,DIM*i+0)]=myresistance[3];
    resistance[myindex(DIM*i+0,DIM*i+2)]=myresistance[4];
    resistance[myindex(DIM*i+2,DIM*i+0)]=myresistance[4];
    resistance[myindex(DIM*i+1,DIM*i+2)]=myresistance[5];
    resistance[myindex(DIM*i+2,DIM*i+1)]=myresistance[5];
  }
}


// this adds the identity matrix to a given matrix of ld=size
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
// block : (ignored) the number of elements to process per thread
//         if this is e.g. 3 and the matrix is 3Nx3N, than N threads have to be started
__global__ void sd_add_identity_matrix(double * matrix, int size, int lda){
  //int lda=((size+31)/32)*32;
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  //for (int i = idx*block; i< (idx+1)*block; i++){
  for (int i = idx;i< size; i+=blockDim.x*gridDim.x){
    matrix[i+i*lda]+=1;
  }
}

// this sets a block to zero
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
__global__ void sd_set_zero_matrix(double * matrix, int size){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int matsize=((size+31)/32)*32;
  matsize*=size;
  for (int i = idx;i< matsize; i+=blockDim.x*gridDim.x){
    matrix[i]=0;
  }
}


// this sets a block to zero
// data  : pointer to the given data
// size  : the size of the data
__global__ void sd_set_zero(double * data, int size){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  for (int i = idx;i< size; i+=blockDim.x*gridDim.x){
    data[i]=0;
  }
}




// check whether there was any cuda error so far.
// do not use this function directly but use the macro cudaCheckError(const char *msg);
// which requires only the first paramter
// PARAMTERS:
// msg   : the message which should be printed in case of an error
// file  : the file in which the function is called
// line  : the line in which the function is called
void _cudaCheckError(const char *msg, const char * file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err)
    {
      std::cerr <<  "Cuda error:" <<  msg << ": '" <<  hipGetErrorString( err) << "' in "<<file << " l. "<<line<<"\n";
      errexit();
    }
}





#define DIST (2+1e-1)
#define DISP_MAX (10000)

__global__ void sd_real_integrate_prepare( double * r_d , double * disp_d, double * L, double a, int N){
  /*for (int idx = blockIdx.x*blockDim.x + threadIdx.x;
       idx<N ;
       idx+=blockDim.x*gridDim.x){
    double disp2=0;
#pragma unroll
    for (int d=0;d<DIM;d++){
      disp2+=disp_d[idx*DIM+d]*disp_d[idx*DIM+d];
    }
    if (disp2 > DISP_MAX*DISP_MAX){
      double fac=DISP_MAX/sqrt(disp2);
#pragma unroll
      for (int d=0;d<DIM;d++){
	disp_d[idx*DIM+d]*=fac;
      }
    }
  }*/
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  i*=3;
  double disp2;
#pragma unroll
  for (int d=0;d<3;d++){
    disp2+=disp_d[i+d]*disp_d[i+d];
  }
  if (disp2> DISP_MAX*DISP_MAX){
    disp2=DISP_MAX/sqrt(disp2);
#pragma unroll
    for (int d=0;d<3;d++){
      disp_d[i+d]*=disp2;
    }
  }
}
__global__ void sd_real_integrate( double * r_d , double * disp_d, double * L, double a, int N)
{
  
  //for (int idx = blockIdx.x*blockDim.x + threadIdx.x;
  //     idx<N ;
  //     idx+=blockDim.x*gridDim.x){
  int idx =  blockIdx.x*blockDim.x + threadIdx.x;
  // t is the factor how far of disp_d we will move.
  // in case everything is fine, we will move t, if there is some trouble,
  // we will move less to avoid collision
  double t=1;
  double rnew[DIM];
  for (int d=0;d<DIM;d++){
    rnew[d]=r_d[DIM*idx+d]+disp_d[DIM*idx+d];
  }
  const double distmin=(3*a)*(3*a);
  for (int i=0;i<N;i++){
    if (idx==i){
      i++;
      if (i >N){
	continue;
      }
    }
    double dr2=0;
    for (int d=0;d<DIM;d++){
      double tmp=r_d[i*DIM+d]-rnew[d];
      tmp-=L[d]*rint(tmp/L[d]);
      dr2+=tmp*tmp;
    }
    if (dr2 <distmin){ // possible colision - check better
      dr2=0;
      //double dr2o=0; // or do we need old distance?
      for (int d=0;d<DIM;d++){
	double tmp=r_d[i*DIM+d]+disp_d[i*DIM+d]-rnew[d];
	tmp-=L[d]*rint(tmp/L[d]);
	dr2+=tmp*tmp;
	//tmp=r_d[i*DIM+d]-r_d[idx*DIM+d];
	//tmp-=L*rint(tmp/L);
	//dr2o+=tmp*tmp;
      }
      if (dr2 < DIST*DIST*a*a){ // do they collide after the step?
	// ideal: the motion which is responsible for the crash: avoid it.
	// just move them that far that they nearly touch each other.
	// therefore we need the soluten of an quadratic equation
	// in case they are already closer than DIST*a this will move them appart.
	// first: get the coefficents
	double alpha=0,beta=0,gamma=0;
	for (int d=0;d<DIM;d++){
	  double t1=r_d[i*DIM+d]-r_d[idx*DIM+d];
	  t1-=L[d]*rint(t1/L[d]);
	  double t2=disp_d[i*DIM+d]-disp_d[idx*DIM+d];
	  //t2-=L*rint(t2/L); // we would have a problem if we would need to fold back these ...
	  alpha +=t2*t2;
	  beta  +=2*t1*t2;
	  gamma +=t1*t1;
	} 
	// now we want to solve for t: alpha*t**2+beta*t+gamma=DIST*a
	// we want the solution with the minus in the 'mitternachtsformel'
	// because the other solution is when the particles moved through each other
	double tnew = (-beta-sqrt(beta*beta-4*alpha*gamma))/(2*alpha);
	if (tnew < t){ // use the smallest t
	  t=tnew;
	}
      }
    }
  }
  for (int d=0;d<DIM;d++){ // actually do the integration
    r_d[DIM*idx+d]+=disp_d[DIM*idx+d]*t;
  }
  //#warning "Debug is still enabaled"
    //pos_d[DIM*N+idx]=t;
}

__global__ void sd_bucket_sort( double * pos , double * bucketSize, int * bucketNum, int N,
				int * particleCount, int * particleList, int maxParticlePerCell, int totalBucketNum){
  for (int i = blockIdx.x*blockDim.x + threadIdx.x;
       i<N ;
       i+=blockDim.x*gridDim.x){
    int3 bucket;
#pragma unroll 3
    for (int d =0; d<3; d++){
      double tmp;
      // no asm version:
      // tmp = pos[i*3+d];
      // asm version avoids caching
      asm("ld.global.cs.f64 %0,[%1];\n"
	  : "=d"(tmp) : "l"(pos+i*3+d) : );
      tmp/=bucketSize[d];
      int x;
      // this should work - but somehow it does not compile
      //x=__double2int_rd(tmp);
      // the following code is an replacement ...
      // but with this the loop is not getting unrolled
      //asm("cvt.rmi.s32.f64 %0, %1;\n"
      //    : "=r"(x) : "d"(tmp) : );
      // this should also work.
      // but the corresponding ptx code first rounds, and then converts in a second step ...
      // this could lead to rounding errors ...
      x=floor(tmp);
      x%=bucketNum[d];
      // avoid negativ numbers
      x+=bucketNum[d];
      x%=bucketNum[d];
      switch (d){
      case 0:
	bucket.x = x;
	break;
      case 1:
	bucket.y = x;
	break;
      case 2:
	bucket.z = x;
	break;
      }
    }
    int myBucket = bucket.x + bucket.y*bucketNum[0] + bucket.z*bucketNum[0]*bucketNum[1];
    int num = atomicAdd(particleList+myBucket, 1);
    if (num < maxParticlePerCell){ // every thread should do this - so this is not a branch ...
      particleList[myBucket+num*totalBucketNum]=i;
    }else{
      // Note: printf in device code works only with cc>=2.0 //
#if (__CUDA_ARCH__>=200)
      printf("error: overflow in grid cell (%i,%i,%i)\n",bucket.x,bucket.y,bucket.z);
#endif
    }
  }
}






#endif
