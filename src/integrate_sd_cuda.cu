#include "hip/hip_runtime.h"
/*
   Copyright (C) 2010,2011,2012 The ESPResSo project

   This file is part of ESPResSo.

   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "config.hpp"
#ifdef CUDA /* Terminates at end of file */

/* This is where the hydro dynamic interaction is implemented */
//#include "inversion/GPUGausSeidel.h"
//#include "integrate_sd_cuda.cuh"



#include <stdio.h>
#include <iostream>

#include "assert.h"
#include "integrate_sd_cuda_debug.cuh"
#include "integrate_sd.hpp" // this includes magma and cublas
#include "cuda_utils.hpp"
#include "errorhandling.hpp"
#include "global.hpp"

const int numThreadsPerBlock = 128;

void _cudaCheckError(const char *msg, const char * file, const int line);
#define cudaCheckError(msg)  _cudaCheckError((msg),__FILE__,__LINE__)

#define myindex(i,j,N) ((i)*(DIM*(N))+(j))


/* ************************************* *
 * *******   private functions   ******* *
 * ************************************* */
void sd_compute_mobility(hipblasHandle_t cublas, double * r_d, int N, double eta, double a, double * L_d, double * total_mobility_d);

// This computes the farfield contribution.
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// mobility is the mobility matrix which will be retruned
// L is the boxlength
__global__ void sd_compute_mobility_matrix(double * r, int N, double self_mobility, double a, double * L, double * mobility);


__global__ void sd_add_identity_matrix(double * matrix, int size, int block);
void _cudaCheckError(const char *msg, const char * file, const int line);
// this computes the near field
// it calculates the ResistanceMatrix
__global__ void sd_compute_resistance_matrix(double * r, int N, double self_mobility, double a, double * L, double * resistance);


__global__ void sd_real_integrate( double * r_d , double * disp_d, double * L, double a, int N);


// this sets a block to zero
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
__global__ void sd_set_zero_matrix(double * matrix, int size);


/* ************************************* *
 * *******     implementation    ******* *
 * ************************************* */

// this calls all the functions to:
//  * generate the mobility matrix (farfield and nearfield)
//  * compute the displacements
//  * add the displacements to the positions
// TODO: add brownian motion, which is current missing
// PARAMTERS:
// box_l_h : the size of the box in x,y and z-direction, on the host (in)
// N       : Number of particles (in)
// pos_h   : position of the particles, simple* array on host (in and out)
// force_h : forces on the particles, simple* array on host (in)
// * : a simple array is e.g. [x_1, y_1, z_1, x_2, y_2, z_2, ...]
void propagate_pos_sd_cuda(double * box_l_h, int N,double * pos_h, double * force_h, double * velo_h){
  //printVectorHost(pos_h,3*N,"pos after call");
  double viscosity=sd_viscosity;
  double radius   =sd_radius;
  if (viscosity  < 0){
    std::cerr << "The viscosity for SD was not set\n";
    errexit();
  }
  if (radius  < 0){
    std::cerr << "The particle radius for SD was not set\n";
    errexit();
  }
  
  static hipblasHandle_t cublas=NULL;
  if (cublas==NULL){
    if (hipblasCreate(&cublas) != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "CUBLAS initialization failed\n";
      errexit();
    }
    magma_init();
  }

  double * box_l_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&box_l_d, 3*sizeof(double)));
  cuda_safe_mem(hipMemcpy(box_l_d,box_l_h,3*sizeof(double),hipMemcpyHostToDevice));
  double * pos_d=NULL;
  //#warning debug: pos is to large ...
  cuda_safe_mem(hipMalloc((void**)&pos_d, (DIM)*N*sizeof(double)));
  cuda_safe_mem(hipMemcpy(pos_d,pos_h,N*DIM*sizeof(double),hipMemcpyHostToDevice));
  //printVectorDev(pos_d,3*N,"pos after copy");
  double * force_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&force_d, DIM*N*sizeof(double)));
  cuda_safe_mem(hipMemcpy(force_d,force_h,N*DIM*sizeof(double),hipMemcpyHostToDevice));
  double * mobility_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&mobility_d, DIM*DIM*N*N*sizeof(double)));
  double * disp_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&disp_d, DIM*N*sizeof(double)));
  
  sd_compute_mobility(cublas, pos_d, N, viscosity, radius, box_l_d, mobility_d);
  
  
  //double alpha=1;
  double beta=0;
  hipblasStatus_t stat = hipblasDgemv( cublas, HIPBLAS_OP_T, DIM*N, DIM*N, &time_step, mobility_d, DIM*N, force_d, 1, &beta, disp_d, 1);
  if (stat != HIPBLAS_STATUS_SUCCESS) { std::cerr << "CUBLAS Multiplication failed!\n"; errexit(); }
  
  //int numThreadsPerBlock = 3;
  int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;
  //stat = hipblasDaxpy(cublas, DIM*N, &alpha, v_d, 1, xr_d, 1);
  //assert(stat==HIPBLAS_STATUS_SUCCESS);
  sd_real_integrate<<< numBlocks , numThreadsPerBlock  >>>(pos_d , disp_d, box_l_d, sd_radius, N);
  
  // copy back the positions
  cuda_safe_mem(hipMemcpy(pos_h,pos_d,N*DIM*sizeof(double),hipMemcpyDeviceToHost));
  // save the displacements as velocities (maybe somebody is interested)
  double alpha=1/time_step;
  stat = hipblasDscal(cublas, DIM*N, &alpha, disp_d, 1);
  assert(stat == HIPBLAS_STATUS_SUCCESS);
  cuda_safe_mem(hipMemcpy(velo_h,disp_d,N*DIM*sizeof(double),hipMemcpyDeviceToHost));
  

  cuda_safe_mem(hipFree((void*)box_l_d));
  cuda_safe_mem(hipFree((void*)pos_d));
  cuda_safe_mem(hipFree((void*)force_d));
  cuda_safe_mem(hipFree((void*)mobility_d));
  cuda_safe_mem(hipFree((void*)disp_d));
}



// calculate the farfield and the nearfield and add them
// PARAMETERS:
// cublas : a valid handle of cublas (in)
// r_d    : position of the particles on the device, size 3*N (in)
//          the form has to be [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N      : Number of particles (in)
// eta    : viscositiy of the fluid (in)
// a      : Particle radius (in)
// L_d    : boxsize in x y and z-directions (in)
// total_mobility_d: matrix of the computed total mobility, size 3*3*N*N (in/out, is overwritten)
void sd_compute_mobility(hipblasHandle_t cublas, double * r_d, int N, double eta, double a, double * L_d, double * total_mobility_d){
  hipDeviceSynchronize(); // just for debugging
  cudaCheckError("");
  //int numThreadsPerBlock = 32;
  int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;
  
  // compute the mobility Matrix
  double * helper_d=NULL;
  int ressize=max(4000,DIM*DIM*N*N); // ressize has to be larger for small matrizes (else magma complains)
  cuda_safe_mem(hipMalloc( (void**)&helper_d, ressize*sizeof(double) ));
  assert(helper_d);
  double * mobility_d=NULL;
  cuda_safe_mem(hipMalloc( (void**)&mobility_d, DIM*DIM*N*N*sizeof(double) ));
  assert(mobility_d);
  //printMatrixDev(mobility_d,3*N,3*N,"before mobility:");
  //printVectorDev(r_d,3*N,"positions");
  sd_set_zero_matrix<<<numBlocks, numThreadsPerBlock >>>(mobility_d,3*N);
  sd_compute_mobility_matrix<<< numBlocks , numThreadsPerBlock  >>>(r_d,N,1./(6.*M_PI*eta*a), a, L_d, mobility_d);
  hipDeviceSynchronize(); // just for debugging
  printMatrixDev(mobility_d,3*N,3*N,"mobility_d");
  cudaCheckError("compute mobility error");
  //printMatrixDev(mobility_d,3*N,3*N,"early mobility:");
  // compute the resistance matrix
  double * resistance_d=NULL;
  cuda_safe_mem(hipMalloc( (void**)&resistance_d, ressize*sizeof(double) )); //this needs to be bigger for matrix inversion
  assert(resistance_d !=NULL);
  sd_set_zero_matrix<<<numBlocks, numThreadsPerBlock >>>(resistance_d,3*N);
  sd_compute_resistance_matrix<<< numBlocks , numThreadsPerBlock  >>>(r_d,N,1./(6.*M_PI*eta*a), a, L_d, resistance_d);
  hipDeviceSynchronize(); // we need both matrices to continue;
  cudaCheckError("compute resistance or mobility error");
  hipblasStatus_t status;
  
  //debug
  //printMatrixDev(mobility_d,3*N,3*N,"late mobility:");
  //printVectorDev(r_d,3*N,"position: ");
  //printMatrixDev(resistance_d,3*N,3*N,"resitstance: ");
  
  double alpha=1, beta =0;
  status = hipblasDgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, DIM*N , DIM*N ,DIM*N, &alpha, mobility_d, DIM*N,resistance_d, DIM*N, &beta,helper_d, DIM*N);
  assert(status == HIPBLAS_STATUS_SUCCESS);
  sd_add_identity_matrix<<< numBlocks , numThreadsPerBlock  >>>(helper_d,DIM*N,DIM);
  hipDeviceSynchronize();
  
  cudaCheckError("add identity error");
  
  // inverting the matrix 
  int ipiv[DIM*N];
  int info;
  magma_dgetrf_gpu( DIM*N, DIM*N,helper_d, DIM*N, ipiv, &info);
  assert(info==0);
  magma_dgetri_gpu( N*DIM, helper_d, DIM*N, ipiv,resistance_d,ressize, &info);
  assert(info==0);
  // compute the inverse matrix
  // this is an alternative implementation ...
  // be sure to make sure everything else matches, because this one does not overwrite the original matrix
  // GPUGausSeidelDev(helper_d,inverse_d,DIM*N);
  hipDeviceSynchronize();
  cudaCheckError("inversion error");
  // compute total_mobility_d
  status = hipblasDgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, DIM*N , DIM*N ,DIM*N, &alpha, \
		       mobility_d, DIM*N,helper_d, DIM*N, &beta,total_mobility_d, DIM*N);
  assert(status == HIPBLAS_STATUS_SUCCESS);
  // free the two matrices again
  hipFree((void*)resistance_d);
  hipFree((void*)mobility_d);
  hipFree((void*)helper_d);
  cudaCheckError("in mobility");
}


// This computes the farfield contribution of the mobility
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// mobility is the mobility matrix which will be retruned
// L_d is the boxlength
__global__ void sd_compute_mobility_matrix(double * r, int N, double self_mobility, double a, double * L_g, double * mobility){
  double mypos[3];
  __shared__ double L[3];
  __shared__ double cachedPos[3*numThreadsPerBlock];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < 3){ // copy L to shared memory
    L[threadIdx.x]=L_g[threadIdx.x];
  }
  __syncthreads();
  // get data for myposition - using coalscaled memory access
  for (int j=0;j<3;j++){
    cachedPos[numThreadsPerBlock*j+threadIdx.x] = r[numThreadsPerBlock*j+i];
  }
  __syncthreads();
  for (int j=0;j<3;j++){
    mypos[j] = cachedPos[threadIdx.x*3+j];
  }

  if (i < N){
    // first write the self contribution
#pragma unroll
    for (int k=0; k < DIM; k++){
      //#pragma unroll
      //for (int l=0; l < DIM; l++){
      //mobility[myindex(DIM*i+k,DIM*i+l,N)]=0;
      //}
      mobility[myindex(DIM*i+k,DIM*i+k,N)]=self_mobility;
    }
  }
  for (int offset=0;offset<N;offset+=numThreadsPerBlock){
    // copy positions to shared memory
#pragma unroll
    for (int j=0;j<3;j++){
      cachedPos[threadIdx.x+j*numThreadsPerBlock] = r[numThreadsPerBlock*j+i];
    }
    __syncthreads();
    if (i < N){
      for (int j=offset;j<offset+numThreadsPerBlock;j++){
	if (i==j){
	  j++; //just continue with next particle
	  if (j==offset+numThreadsPerBlock){
	    continue;
	  }
	}
	if (j < N){
	  double dr[DIM];
	  double dr2=0;
#pragma unroll 3
	  for (int k=0;k<DIM;k++){
	    dr[k]=r[DIM*i+k]-r[DIM*j+k]; // r_ij
	    //dr[k]=mypos[k]-cachedPos[DIM*(j-offset)+k]; // r_ij
	    /*if (isnan(dr[k])){
	      dr[k]=1337;
	      }*/
	    dr[k]-=rint(dr[k]/L[k])*L[k]; // fold back
	    dr2+=dr[k]*dr[k];
	  }
	  if (dr2 < 0.1){
	    dr2=0.1;
	  }
	  double drn= sqrt(dr2); // length of dr
	  double b = a/drn;
      
	  if (0.5 < b){  // drn < 2*a
	    /*double t=3./32./drn/a*self_mobility;
	      double t2=(1-9./32.*drn/a)*self_mobility;
	      for (k=0; k < DIM; k++){
	      for (l=0;l < DIM; l++){
	      mobility[myindex(DIM*i+k,DIM*j+l,N)]=dr[k]*dr[l]*t;
	      }
	      mobility[myindex(DIM*i+k,DIM*j+k,N)]+=t2;
	      }*/ // this should not happen ...
	    // python implementation:
	    //T=one*(1-9./32.*drn/a)+3./32.*dr*drt/drn/a;
	  }
	  else{
	    double b2=b*b;
	    double t=(0.75-1.5*b2)*b/dr2*self_mobility;
	    double t2=(0.75+0.5*b2)*b*self_mobility;
	    /*if (isnan(t)){
	      t=1337;
	      }
	      if (isnan(t2)){
	    t2=1337;
	    }*/
#pragma unroll
	    for (int k=0; k < DIM; k++){
#pragma unroll
	      for (int l=0;l < DIM; l++){
		mobility[myindex(DIM*i+k,DIM*j+l,N)]=dr[k]*dr[l]*t;
	      }
	      mobility[myindex(DIM*i+k,DIM*j+k,N)]+=t2;
	    }
	    // python implementation:
	    // T=one*(0.75+0.5*b2)*b+(0.75-1.5*b2)*b*drt*dr/dr2;
	  }
	}
      }
    }
  }
}
  
// this computes the near field
// it calculates the ResistanceMatrix
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// L_d is the boxlength
// resistance is the resistance matrix which will be retruned
__global__ void sd_compute_resistance_matrix(double * r, int N, double self_mobility, double a, double * L_g, double * resistance){
  double mypos[3];
  __shared__ double L[3];
  __shared__ double cachedPos[3*numThreadsPerBlock];
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < 3){ // copy L to shared memory
    L[threadIdx.x]=L_g[threadIdx.x];
  }
  __syncthreads();
  // get data for myposition - but alligned
  for (int j=0;j<3;j++){
    cachedPos[threadIdx.x+j*numThreadsPerBlock] = r[idx+j*numThreadsPerBlock];
  }
  __syncthreads();
  for (int j=0;j<3;j++){
    mypos[j] = cachedPos[threadIdx.x*3+j];
  }
  
  
  
  
  for (int i = idx; i < N; i+=blockDim.x*gridDim.x){
#pragma unroll 3
    for (int k=0; k < DIM; k++){
#pragma unroll 3
      for (int l=0;l < DIM; l++){
	resistance[myindex(DIM*i+k,DIM*i+l,N)]=0; // we will add some terms on the diagonal, so set it to zero before
      }
    }
    for (int offset=0;offset<N;offset+=numThreadsPerBlock){
      // copy positions to shared memory
#pragma unroll
      for (int j=0;j<3;j++){
	cachedPos[threadIdx.x+j*numThreadsPerBlock] = r[numThreadsPerBlock*j+i];
      }
      __syncthreads();
      for (int j=offset;j<offset+numThreadsPerBlock;j++){
	
	//for (int j=0;j<N;j++){
	if (i==j){ // skip self contribution
	  j++;
	  if (j==offset+numThreadsPerBlock){
	    continue;
	  }
	}
	double dr[DIM];
	double dr2=0;
#pragma unroll
	for (int k=0;k<DIM;k++){
	  dr[k]=mypos[k]-cachedPos[3*(j-offset)+k]; // r_ij
	  dr[k]-=L[k]*rint(dr[k]/L[k]); // fold back
	  dr2+=dr[k]*dr[k];
	}
	if (dr2 < 4*a*4*a && 2*a*2*a < dr2 ){// check whether 2*a < drn < 4*a
	  // python code:
	  // # Use only singular therms, namely to order O(s_ij^0)                                                                  
	  // T=(1./4./s-1/4-9./40.*ls)*dr*drt/dr2
	  // #           ^ this additonal constant is so that the mobility is smooth
	  // # c.f. N.-Q. Nguyen and A. J. C. Ladd, PHYSICAL REVIEW E 66, 046708 (2002) equation (34)                               
	  // T+=1./6.*ls*(-one+dr*drt/dr2)
	  // R[3*i:3*i+3,3*j:3*j+3]=-T
	  // R[3*i:3*i+3,3*i:3*i+3]+=T
	  double drn= sqrt(dr2); // length of dr
	  double s = drn/a-2;
	  double ls = log(s);
	  
	  double const t_c=-0.125+9./40.*log(2.)+3./112.*2.*log(2.);
	  double const t2_c=2./6.*log(2.);
	  double t=(-0.25/s+9./40.*ls+3./112.*s*ls-t_c)/dr2/self_mobility;
	  double t2=(1./6.*ls-t2_c)/self_mobility;
#pragma unroll 3
	  for (int k=0; k < DIM; k++){
#pragma unroll 3
	    for (int l=0;l < DIM; l++){
	      resistance[myindex(DIM*i+k,DIM*j+l,N)]=dr[k]*dr[l]*t;
	      resistance[myindex(DIM*i+k,DIM*i+l,N)]-=dr[k]*dr[l]*t;
	    }
	    resistance[myindex(DIM*i+k,DIM*j+k,N)]+=t2;
	    resistance[myindex(DIM*i+k,DIM*i+k,N)]-=t2;
	  }
	  // python implementation:
	  //T=one*(1-9./32.*drn/a)+3./32.*dr*drt/drn/a;
	}
	else{ // set the block to zero
	  // it might be faster to set everything in the beginning to zero ...
	  // or use sparse matrices ...
#pragma unroll 3
	  for (int k=0; k < DIM; k++){
#pragma unroll 3
	    for (int l=0;l < DIM; l++){
	      resistance[myindex(DIM*i+k,DIM*j+l,N)]=0;
	    }
	  }  
	}
      }
    }
  }
}


// this adds the identity matrix to a given matrix of ld=size
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
// block : the number of elements to process per thread
//         if this is e.g. 3 and the matrix is 3Nx3N, than N threads have to be started
__global__ void sd_add_identity_matrix(double * matrix, int size, int block){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  //for (int i = idx*block; i< (idx+1)*block; i++){
  for (int i = idx;i< size; i+=blockDim.x*gridDim.x){
    //#define myindex(i,j,N) ((i)*(DIM*(N))+(j))
    if ( i < size)
      matrix[i+i*size]+=1;
  }
}

// this sets a block to zero
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
__global__ void sd_set_zero_matrix(double * matrix, int size){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  //for (int i = idx*block; i< (idx+1)*block; i++){
  for (int i = idx;i< size*size; i+=blockDim.x*gridDim.x){
    //#define myindex(i,j,N) ((i)*(DIM*(N))+(j))
    //if ( i < size)
    matrix[i]=0;
  }
}






// check whether there was any cuda error so far.
// do not use this function directly but use the macro cudaCheckError(const char *msg);
// which requires only the first paramter
// PARAMTERS:
// msg   : the message which should be printed in case of an error
// file  : the file in which the function is called
// line  : the line in which the function is called
void _cudaCheckError(const char *msg, const char * file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err)
    {
      std::cerr <<  "Cuda error:" <<  msg << ": '" <<  hipGetErrorString( err) << "' in "<<file << " l. "<<line<<"\n";
      errexit();
    }
}





#define DIST (2+1e-1)
#define DISP_MAX (0.5)

__global__ void sd_real_integrate( double * r_d , double * disp_d, double * L, double a, int N)
{
  
  for (int idx = blockIdx.x*blockDim.x + threadIdx.x;
       idx<N ;
       idx+=blockDim.x*gridDim.x){
    // t is the factor how far of disp_d we will move.
    // in case everything is fine, we will move t, if there is some trouble,
    // we will move less to avoid collision
    double t=1;
    double disp2=0;
    //TODO: FIXME: put to separate kernel to avoid race condition
    for (int d=0;d<DIM;d++){
      disp2+=disp_d[idx*DIM+d]*disp_d[idx*DIM+d];
    }
    if (disp2 > DISP_MAX*DISP_MAX){
      double fac=DISP_MAX/sqrt(disp2);
      for (int d=0;d<DIM;d++){
	disp_d[idx*DIM+d]*=fac;
      }
    }
    double rnew[DIM];
    for (int d=0;d<DIM;d++){
      rnew[d]=r_d[DIM*idx+d]+disp_d[DIM*idx+d];
    }
    const double distmin=(3*a)*(3*a);
    for (int i=0;i<N;i++){
      if (idx==i)
	continue;
      double dr2=0;
      for (int d=0;d<DIM;d++){
	double tmp=r_d[i*DIM+d]-rnew[d];
	tmp-=L[d]*rint(tmp/L[d]);
	dr2+=tmp*tmp;
      }
      if (dr2 <distmin){ // possible colision - check better
	dr2=0;
	//double dr2o=0; // or do we need old distance?
	for (int d=0;d<DIM;d++){
	  double tmp=r_d[i*DIM+d]+disp_d[i*DIM+d]-rnew[d];
	  tmp-=L[d]*rint(tmp/L[d]);
	  dr2+=tmp*tmp;
	  //tmp=r_d[i*DIM+d]-r_d[idx*DIM+d];
	  //tmp-=L*rint(tmp/L);
	  //dr2o+=tmp*tmp;
	}
	if (dr2 < DIST*DIST*a*a){ // do they collide after the step?
	  // ideal: the motion which is responsible for the crash: avoid it.
	  // just move them that far that they nearly touch each other.
	  // therefore we need the soluten of an quadratic equation
	  // in case they are already closer than DIST*a this will move them appart.
	  // first: get the coefficents
	  double alpha=0,beta=0,gamma=0;
	  for (int d=0;d<DIM;d++){
	    double t1=r_d[i*DIM+d]-r_d[idx*DIM+d];
	    t1-=L[d]*rint(t1/L[d]);
	    double t2=disp_d[i*DIM+d]-disp_d[idx*DIM+d];
	    //t2-=L*rint(t2/L); // we would have a problem if we would need to fold back these ...
	    alpha +=t2*t2;
	    beta  +=2*t1*t2;
	    gamma +=t1*t1;
	  } 
	  // now we want to solve for t: alpha*t**2+beta*t+gamma=DIST*a
	  // we want the solution with the minus in the 'mitternachtsformel'
	  // because the other solution is when the particles moved through each other
	  double tnew = (-beta-sqrt(beta*beta-4*alpha*gamma))/(2*alpha);
	  if (tnew < t){ // use the smallest t
	    t=tnew;
	  }
	}
      }
    }
    for (int d=0;d<DIM;d++){ // actually do the integration
      r_d[DIM*idx+d]+=disp_d[DIM*idx+d]*t;
    }
    //#warning "Debug is still enabaled"
    //pos_d[DIM*N+idx]=t;
  }
}








#endif
