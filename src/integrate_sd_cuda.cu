#include "hip/hip_runtime.h"
/*
   Copyright (C) 2010,2011,2012 The ESPResSo project

   This file is part of ESPResSo.

   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.

   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.

   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "config.hpp"
#ifdef CUDA /* Terminates at end of file */

/* This is where the hydro dynamic interaction is implemented */

// TODO:
// * use preconditioner in iterative solver
// * implement matrix-free farfield (via fft)
// * add brownian motion
// * add bucket versions


#include <stdio.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include <hip/device_functions.h>
// C++98:
#include <algorithm>
// C++11:
//#include <utility>

#include "assert.h"
#include "integrate_sd_cuda_debug.cuh"
#include "integrate_sd.hpp" // this includes magma and cublas
#include "cuda_utils.hpp"
#include "errorhandling.hpp"
#include "global.hpp"

#ifndef SD_USE_FLOAT
#warning using double
typedef double real;
#define cublasRgemm(...)              hipblasDgemm( __VA_ARGS__)
#define cublasRdot(...)               hipblasDdot( __VA_ARGS__)
#define cublasRgemv(...)              hipblasDgemv( __VA_ARGS__)
#define cublasRcopy(...)              hipblasDcopy( __VA_ARGS__)
#define cublasRaxpy(...)              hipblasDaxpy( __VA_ARGS__)
#define cublasRscal(...)              hipblasDscal( __VA_ARGS__)
#define cublasRnrm2(...)              hipblasDnrm2( __VA_ARGS__)
#define curandGenerateNormalReal(...) hiprandGenerateNormalDouble(__VA_ARGS__)
#define __real2int_rd(...)            __double2int_rd(__VA_ARGS__)
#define rnaupd(...)                   dnaupd_(__VA_ARGS__)
#else // use float
#warning Using Float
typedef float real;
#define cublasRgemm(...)              hipblasSgemm( __VA_ARGS__)
#define cublasRdot(...)               hipblasSdot( __VA_ARGS__)
#define cublasRgemv(...)              hipblasSgemv( __VA_ARGS__)
#define cublasRcopy(...)              hipblasScopy( __VA_ARGS__)
#define cublasRaxpy(...)              hipblasSaxpy( __VA_ARGS__)
#define cublasRscal(...)              hipblasSscal( __VA_ARGS__)
#define cublasRnrm2(...)              hipblasSnrm2( __VA_ARGS__)
#define curandGenerateNormalReal(...) hiprandGenerateNormal(__VA_ARGS__)
#define __real2int_rd(...)            __float2int_rd(__VA_ARGS__)
#define rnaupd(...)                   snaupd_(__VA_ARGS__)
#endif //#ifndef SD_USE_FLOAT


extern double temperature; // this is defined in thermostat.cpp

const int numThreadsPerBlock = 32;

void _cudaCheckError(const char *msg, const char * file, const int line);
#define cudaCheckError(msg)  _cudaCheckError((msg),__FILE__,__LINE__)

#define myindex(i,j) ((i)*(lda)+(j))

#define SQR(x) (x)*(x)

#define cublasCall(call) { hipblasStatus_t stat=(call);	\
    assert(stat==HIPBLAS_STATUS_SUCCESS);		\
  }
#define curandCall(call) { hiprandStatus_t stat =(call);	\
    assert(stat == HIPRAND_STATUS_SUCCESS);		\
  }

// headers for ARPACK-NG: http://forge.scilab.org/index.php/p/arpack-ng/
extern "C"
{
  void dnaupd_(int* IDO, char* BMAT, int* N, char WHICH[], int* NEV, double* TOL, double RESID[], int* NCV, double V[], int* LDV, int IPARAM[],
	       int IPNTR[], double WORKD[], double WORKL[], int* LWORKL, int* INFO);
  void snaupd_(int* IDO, char* BMAT, int* N, char WHICH[], int* NEV, float* TOL, float RESID[], int* NCV, float V[], int* LDV, int IPARAM[],
	       int IPNTR[], float WORKD[], float WORKL[], int* LWORKL, int* INFO);
  // this is to compute eigenvectors - but we want only eigenvalues
  //void dneupd_(int* RVEC, char* HOWMNY, int SELECT[], double DR[], double DI[], double Z[], int* LDZ, double* SIGMAR, double* SIGMAI, double WORKEV[],
  //	       char* BMAT, int* N, char WHICH[], int* NEV, double* TOL, double RESID[], int* NCV, double V[], int* LDV, int IPARAM[], int IPNTR[],
  //	       double WORKD[], double WORKL[], int* LWORKL, int* INFO);
}

#define SD_RESISTANCE_CORRECT

/* ************************************* *
 * *******   private functions   ******* *
 * ************************************* */
void sd_compute_displacement(hipblasHandle_t cublas, real * r_d, int N, real eta, real a, real * L_d, 
			     real * total_mobility_d, real * force_d, real * disp_d, int * myInfo);


// this solves iteratively using CG
// disp * (1+resistance*mobility) = mobility_d *  force_d 
// and returnes disp
// mobility and resistance are square matrizes with size <size> and lda <((size+31)/32)*32>
// force and disp are vectors of size <size>
real sd_iterative_solver(hipblasHandle_t cublas, const real * mobility, const real * resistance, const real * force, int size, real * disp);

// This computes the farfield contribution.
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// mobility is the mobility matrix which will be retruned
// L is the boxlength
__global__ void sd_compute_mobility_matrix(real * r, int N, real self_mobility, real a, real * L, real * mobility);

// adds to each of the diagonal elemnts of the sizse*size matrix matrix
// with lda lda 1
__global__ void sd_add_identity_matrix(real * matrix, int size, int lda);
void _cudaCheckError(const char *msg, const char * file, const int line);
// this computes the near field
// it calculates the ResistanceMatrix
__global__ void sd_compute_resistance_matrix(real * r, int N, real self_mobility, real a, real * L, real * resistance, int * myInfo);
// TODO: make the order of arguments uniform (and logical?)
// TODO: description here
__global__ void sd_compute_brownian_force_nearfield(real * r,real * gaussian_nf,int N,real * L, real a, real self_mobility,real * brownian_force_nf);

// make sure to have one thread per particle
__global__ void sd_real_integrate_prepare( real * r_d , real * disp_d, real * L, real a, int N);
__global__ void sd_real_integrate( real * r_d , real * disp_d, real * L, real a, int N);


// this sets a block to zero
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
__global__ void sd_set_zero_matrix(real * matrix, int size);



// this sets a block to zero
// data  : pointer to the given data
// size  : the size of the data
__global__ void sd_set_zero(real * data, int size);

// this sets a block to zero
// data  : pointer to the given data
// size  : the size of the data
// value : the value written to the data block
__global__ void sd_set_int(int * data, int size, int value);


// implementation of a bucket sort algorithm
// puts all the N particles with given position pos 
// and particle radius a within the periodic boundary 
// conditions of boxSize L_i = bucketSize_i * bucketNum_i
// puts them in the list particleList
// pos                device array of particle position xyz
// bucketSize         device array with the number of buckets in x y and z direction
// bucketNum          device array with the size of a bucket in x y and z direction
// N                  number of particles
// particleCount      device array of the numbers of particles per bucket. must be initalized to zero
// particleList       device array of the partilces in each bucket
// maxParticlePerCell maximum particles per cell
// totalBucketNUm     bucketNum[0]*bucketNum[1]*bucketNum[2] - the total number of buckets
__global__ void sd_bucket_sort( real * pos , real * bucketSize, int * bucketNum, int N,
				int * particleCount, int * particleList, int maxParticlePerCell, int totalBucketNum);

// BICGSTAB-Solver
// implimented as given in `Numerik linearer Gleichungssysteme` by Prof. Dr. Andreas Meister
// this solves A*x=b
// cublas a handle for cublas
// size   the size n of the matrix
// A      the given n*n matrix (in)
// lda    the leading demension of A
// b      the given solution vector (in)
// tol    requested tolerance of the solution
// maxit  maximum number of iterations
// x      the requested solution with an initial guess (in/out)
// returns 0 on success, else error code
int sd_bicgstab_solver(hipblasHandle_t cublas ,int size, real * A,int lda, real * b, real tol, int maxit, real * x, real * res);



// calculates the largest and snalles eigenvalue of the matrix
// size        : size of the eigenvector / the matrix           (IN)
// mobility_d  : handle of the mobility matrix (on the device)  (IN)
// lambda_min  : smalles eigenvalue                            (OUT)
// lambda_max  : largest eigenvalue                            (OUT)
void calculate_maxmin_eigenvalues(int size,real *mobility_d,real * lamba_min,real * lambda_max);


// this function should be fast, as the data should fit (mostly) in L1
// lambda_min   : the lower boundery
// lambda_max   : the upper boundery of the interval
// tol          : the given tollerance which should be achieved
// coefficents  : the pointer where the data will be stored
real calculate_chebyshev_coefficents(real lambda_min, real lambda_max, real tol,real ** coefficents);

typedef unsigned long long ull;
// atomicAdd implementation for double
__device__ double atomicAdd(double * address, double inc);

// global variables for usage in this file
hipblasHandle_t cublas=NULL;
/* *************************************************************************************************************** *
 * ********************************************     implementation    ******************************************** *
 * *************************************************************************************************************** */
/* *************************************************************************************************************** *
 * *******     III MM   MM PPP  L     EEEE MM   MM EEEE NN    N TTTTTTT  AAA  TTTTTTT III  OOO  NN    N    ******* *
 * *******      I  M M M M P  P L     E    M M M M E    N N   N    T    A   A    T     I  O   O N N   N    ******* *
 * *******      I  M  M  M PPP  L     EEE  M  M  M EEE  N  N  N    T    AAAAA    T     I  O   O N  N  N    ******* *
 * *******      I  M     M P    L     E    M     M E    N   N N    T    A   A    T     I  O   O N   N N    ******* *
 * *******     III M     M P    LLLL  EEEE M     M EEEE N    NN    T    A   A    T    III  OOO  N    NN    ******* *
 * *************************************************************************************************************** */
/* *************************************************************************************************************** */

/* *************************************************************************************************************** *
 * ********************************************     HOST-Functions    ******************************************** *
 * *************************************************************************************************************** */

// this calls all the functions to:
//  * generate the mobility matrix (farfield and nearfield)
//  * compute the displacements
//  * add the displacements to the positions
// TODO: add brownian motion, which is currently missing
// PARAMTERS:
// box_l_h : the size of the box in x,y and z-direction, on the host (in)
// N       : Number of particles (in)
// pos_h   : position of the particles, simple* array on host (in and out)
// force_h : forces on the particles, simple* array on host (in)
// velo_h  : velocities of the particles, simple* array on host (in and out)
// * : a simple array is e.g. [x_1, y_1, z_1, x_2, y_2, z_2, ...]
void propagate_pos_sd_cuda(real * box_l_h, int N,real * pos_h, real * force_h, real * velo_h){
  //printVectorHost(pos_h,3*N,"pos after call");
  real viscosity=sd_viscosity;
  real radius   =sd_radius;
  if (viscosity  < 0){
    std::cerr << "The viscosity for SD was not set\n";
    errexit();
  }
  if (radius  < 0){
    std::cerr << "The particle radius for SD was not set\n";
    errexit();
  }
  if (time_step < 0){
    std::cerr << "The timestep was not set\n";
    errexit();
  }
  
  int lda=((3*N+31)/32)*32;
  
  //static hipblasHandle_t cublas=NULL;
  if (cublas==NULL){
    hipblasStatus_t stat = hipblasCreate(&cublas);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      std::cerr << "CUBLAS initialization failed in " << __FILE__ << " l. " << __LINE__ <<"\n\t"  ;
      if (stat == HIPBLAS_STATUS_NOT_INITIALIZED){
	std::cerr << "the CUDA Runtime initialization failed.\n";
      } else if (stat == HIPBLAS_STATUS_ALLOC_FAILED) {
	std::cerr << "the resources could not be allocated\n";
      } else {
	std::cerr << "unknown error\n";
      }
      errexit();
    }
    //magma_init();
  }

  real * box_l_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&box_l_d, 3*sizeof(real)));
  cuda_safe_mem(hipMemcpy(box_l_d,box_l_h,3*sizeof(real),hipMemcpyHostToDevice));
  real * pos_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&pos_d, (DIM)*N*sizeof(real)));
  cuda_safe_mem(hipMemcpy(pos_d,pos_h,N*DIM*sizeof(real),hipMemcpyHostToDevice));
  //printVectorDev(pos_d,3*N,"pos after copy");
  real * force_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&force_d, DIM*N*sizeof(real)));
  cuda_safe_mem(hipMemcpy(force_d,force_h,N*DIM*sizeof(real),hipMemcpyHostToDevice));
  real * mobility_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&mobility_d, lda*N*3*sizeof(real)));
  real * disp_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&disp_d, DIM*N*sizeof(real)));
  cuda_safe_mem(hipMemcpy(disp_d,velo_h,N*DIM*sizeof(real),hipMemcpyHostToDevice));
  int myInfo_h[]={0,0,0};
  int * myInfo_d=NULL;
  cuda_safe_mem(hipMalloc((void**)&myInfo_d, 3*sizeof(int)));
  cuda_safe_mem(hipMemcpy(myInfo_d,myInfo_h,3*sizeof(int),hipMemcpyHostToDevice));
  // rescale forces - this should not be done somewhere else ...
  real alpha=time_step;
  cublasCall(cublasRscal( cublas, 3*N, &alpha, force_d, 1));
  //alpha=1/time_step;
  //cublasCall(cublasRscal(cublas, DIM*N, &alpha, disp_d, 1));  
  sd_compute_displacement(cublas, pos_d, N, viscosity, radius, box_l_d, mobility_d, force_d, disp_d, myInfo_d);
  cuda_safe_mem(hipMemcpy(myInfo_h,myInfo_d,3*sizeof(int),hipMemcpyDeviceToHost));
  //std::cerr <<"MyInfo: "<< myInfo_h[0] <<"\t" << myInfo_h[1] <<"\t" << myInfo_h[2] <<"\n";
  
  //int numThreadsPerBlock = 3;
  int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;
  //stat = cublasRaxpy(cublas, DIM*N, &alpha, v_d, 1, xr_d, 1);
  //assert(stat==HIPBLAS_STATUS_SUCCESS);
  alpha=time_step;
  cublasCall(cublasRscal( cublas, 3*N, &alpha, disp_d, 1));  
  sd_real_integrate_prepare<<< numBlocks , numThreadsPerBlock  >>>(pos_d , disp_d, box_l_d, sd_radius, N);
  sd_real_integrate<<< numBlocks , numThreadsPerBlock  >>>(pos_d , disp_d, box_l_d, sd_radius, N);
  
  // copy back the positions
  cuda_safe_mem(hipMemcpy(pos_h,pos_d,N*DIM*sizeof(real),hipMemcpyDeviceToHost));
  // save the displacements as velocities (maybe somebody is interested)
  alpha=1/time_step;
  cublasCall(cublasRscal(cublas, DIM*N, &alpha, disp_d, 1));
  cuda_safe_mem(hipMemcpy(velo_h,disp_d,N*DIM*sizeof(real),hipMemcpyDeviceToHost));

  if (myInfo_h[0]){
    ;
    // this needs to be done later, after the data is put where it was in the beginning ...
    //sd_set_particles_apart();
  }
  
  
  cuda_safe_mem(hipFree((void*)box_l_d));
  cuda_safe_mem(hipFree((void*)pos_d));
  cuda_safe_mem(hipFree((void*)force_d));
  cuda_safe_mem(hipFree((void*)mobility_d));
  cuda_safe_mem(hipFree((void*)disp_d));
  cuda_safe_mem(hipFree((void*)myInfo_d));
}



// calculate the farfield and the nearfield and add them
// PARAMETERS:
// cublas : a valid handle of cublas (in)
// r_d    : position of the particles on the device, size 3*N (in)
//          the form has to be [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N      : Number of particles (in)
// eta    : viscositiy of the fluid (in)
// a      : Particle radius (in)
// L_d    : boxsize in x y and z-directions (in)
// total_mobility_d: matrix of the computed total mobility, size 3*3*N*N (in/out, is overwritten)
void sd_compute_displacement(hipblasHandle_t cublas, real * r_d, int N, real eta, real a, real * L_d, 
			     real * total_mobility_d, real * force_d, real * disp_d, int * myInfo_d)
{
  hipDeviceSynchronize(); // just for debugging
  cudaCheckError("START");
  int lda=((3*N+31)/32)*32;
  //int numThreadsPerBlock = 32;
  int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;
  
  // compute the mobility Matrix
  real * helper_d=NULL;
  int ressize=max(4000,lda*DIM*N); // ressize has to be larger for small matrizes (else magma complains)
  cuda_safe_mem(hipMalloc( (void**)&helper_d, ressize*sizeof(real) ));
  assert(helper_d);
  real * mobility_d=NULL;
  cuda_safe_mem(hipMalloc( (void**)&mobility_d, lda*DIM*N*sizeof(real) ));
  assert(mobility_d);
  sd_set_zero_matrix<<<numBlocks, numThreadsPerBlock >>>(mobility_d,3*N);
  hipDeviceSynchronize(); // just for debugging
  cudaCheckError("sd set zero");
  if(N>32){
    printVectorDev(r_d,96,"pos a: ");
    printVectorDev(r_d+96,N*3-96,"pos b: ");
  }
  else{
    printVectorDev(r_d,3*N,"pos: ");
  }
  sd_compute_mobility_matrix<<< numBlocks , numThreadsPerBlock  >>>(r_d,N,1./(6.*M_PI*eta*a), a, L_d, mobility_d);
  hipDeviceSynchronize(); // just for debugging
  cudaCheckError("compute mobility error");
  // compute the resistance matrix
  real * resistance_d=NULL;
  cuda_safe_mem(hipMalloc( (void**)&resistance_d, ressize*sizeof(real) )); //this needs to be bigger for matrix inversion
  assert(resistance_d !=NULL);
  sd_set_zero_matrix<<<numBlocks, numThreadsPerBlock >>>(resistance_d,3*N);
  hipDeviceSynchronize(); // debug
  cudaCheckError("sd_set_zero");
  sd_compute_resistance_matrix<<< numBlocks , numThreadsPerBlock  >>>(r_d,N,1./(6.*M_PI*eta*a), a, L_d, resistance_d, myInfo_d);
  hipDeviceSynchronize(); // we need both matrices to continue;
  cudaCheckError("compute resistance error");
#ifdef SD_DEBUG
  assert(!hasAnyNanDev(mobility_d,N*3*lda));
  assert(!hasAnyNanDev(resistance_d,N*3*lda));
  assert(isSymmetricDev(resistance_d,lda,N*3));
#endif
  
  double err = sd_iterative_solver(cublas, mobility_d, resistance_d, force_d, 3*N,disp_d);
#ifdef SD_DEBUG
  if (hasAnyNanDev(disp_d,N*3)){
    printVectorDev(disp_d,N*3,"disp");
    printVectorDev(force_d,N*3,"force");
    printMatrixDev(resistance_d,lda,N*3,"resistance produces nans?");
  }
  assert(!hasAnyNanDev(disp_d,N*3));
#endif
  
  // brownian part
  if (temperature > 0){
    int myInfo_h[3];
    cuda_safe_mem(hipMemcpy(myInfo_h,myInfo_d,3*sizeof(int),hipMemcpyDeviceToHost));
    int N_ldd = ((N+31)/32)*32;
    int num_of_rands = N_ldd*myInfo_h[2]*2*DIM+N_ldd*DIM;
    
    real * brownian_force_nf = NULL;
    cuda_safe_mem(hipMalloc( (void**)&brownian_force_nf, (3*N)*sizeof(real) ));     assert(brownian_force_nf != NULL);
    real * brownian_force_ff = NULL;
    cuda_safe_mem(hipMalloc( (void**)&brownian_force_ff, (3*N)*sizeof(real) ));     assert(brownian_force_ff != NULL);
    real * gaussian = NULL;
    cuda_safe_mem(hipMalloc( (void**)&gaussian, (num_of_rands)*sizeof(real) ));     assert(gaussian != NULL);
    real * gaussian_ff = gaussian;
    real * gaussian_nf = gaussian+N_ldd*DIM;
    static hiprandGenerator_t generator = NULL;
    static int               sd_random_generator_offset=0;
    if (generator == NULL){
      curandCall(hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT));
      curandCall(hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned long long)('E'+'S'+'P'+'R'+'e'+'s'+'S'+'o')));
      curandCall(hiprandSetGeneratorOrdering( generator, HIPRAND_ORDERING_PSEUDO_BEST));
      curandCall(hiprandSetGeneratorOffset( generator, sd_random_generator_offset));
    }
    //#ifdef FLATNOISE
    // this does not work yet:
    //curandCall(curandGenerateUniformReal(generator, gaussian_d, num_of_rands, 0, sqrt(24.*temperature/time_step)));
    //#else
    curandCall(curandGenerateNormalReal(generator, gaussian, num_of_rands, 0, sqrt(2.*temperature/time_step)));
    //#endif
    if (myInfo_h[2]){
      std::cerr << "Bla!\n";
      int * gamma_index = NULL;
      cuda_safe_mem(hipMalloc( (void**)&gamma_index, (myInfo_h[2]*N)*sizeof(int) ));    assert(gamma_index != NULL);
      sd_set_zero<<<64,192>>>(brownian_force_nf,3*N);
      sd_set_int<<<64,192>>>(gamma_index,myInfo_h[2]*N, -1);
      sd_compute_brownian_force_nearfield<<<numBlocks, numThreadsPerBlock>>>(r_d, gaussian_nf, N, L_d, a,
									     1./(6.*M_PI*eta*a), brownian_force_nf);
    }// end of near field
    static real * cheby_coefficents=NULL;
    static int N_chebyshev;
    static bool recalc_ew = true;
    int size=3*N;
    real lambda_min, lambda_max;
    if (recalc_ew){
      calculate_maxmin_eigenvalues(3*N,mobility_d,&lambda_min, &lambda_max);
      N_chebyshev = calculate_chebyshev_coefficents(lambda_min, lambda_max,1e-3,&cheby_coefficents);
      recalc_ew=false;
    }
    if (lambda_min < 0){
      printMatrixDev(mobility_d,lda,size,"Mobility has negative eigenvalues!\n");
      errexit();
    }
    real * chebyshev_vec_curr, * chebyshev_vec_last, * chebyshev_vec_next;
    real gaussian_ff_norm;
    sd_set_zero<<<192,192>>>(brownian_force_ff,size);
    hipDeviceSynchronize(); // just for debugging
    cudaCheckError("set zero");
    cublasCall(cublasRnrm2(cublas, size, gaussian_ff, 1, &gaussian_ff_norm));
    chebyshev_vec_curr=gaussian_ff;
    cublasCall(cublasRaxpy( cublas, size, cheby_coefficents+0, chebyshev_vec_curr, 1, brownian_force_ff, 1 ));
    printVectorDev(brownian_force_ff, min(15,size), "l. ?458: ");
    //chebyshev_vec_last=chebyshev_vec_curr;
    chebyshev_vec_last=NULL;
    cuda_safe_mem(hipMalloc( (void**)&chebyshev_vec_last, size*sizeof(real) ));    assert(chebyshev_vec_last != NULL);
    chebyshev_vec_next=NULL;
    cuda_safe_mem(hipMalloc( (void**)&chebyshev_vec_next, size*sizeof(real) ));    assert(chebyshev_vec_next != NULL);
    //sd_set_zero<<<192,192>>>(chebyshev_vec_????,size);
    real lambda_minus=lambda_max-lambda_min;
    for (int i=1;i<=N_chebyshev;i++){
      real alpha=2./lambda_minus, beta =0;
      if (i==1){
	alpha=1./lambda_minus;
      }
      cublasCall(cublasRgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mobility_d, lda, chebyshev_vec_curr, 1, &beta, chebyshev_vec_next , 1));
      alpha=-2*(lambda_min+lambda_max)/lambda_minus;
      if (i==1){
	alpha=-(lambda_min+lambda_max)/lambda_minus;
      }
      cublasCall(cublasRaxpy( cublas, size, &alpha, chebyshev_vec_curr, 1, chebyshev_vec_next, 1 ));
      if (i>1){
	alpha=-1;
	cublasCall(cublasRaxpy( cublas, size, &alpha, chebyshev_vec_last, 1, chebyshev_vec_next, 1 ));
      }
      std::swap(chebyshev_vec_curr,chebyshev_vec_next);
      std::swap(chebyshev_vec_last,chebyshev_vec_next);
      cublasCall(cublasRaxpy( cublas, size, cheby_coefficents+i, chebyshev_vec_curr, 1, brownian_force_ff, 1 ));
      real tmp;
      cublasCall(cublasRnrm2(cublas, size, brownian_force_ff, 1 , &tmp));
      fprintf(stderr,"norm brownian force: %e ", tmp);
      cublasCall(cublasRnrm2(cublas, size, chebyshev_vec_last, 1 , &tmp));
      fprintf(stderr,"norm cheby last: %e ", tmp);
      cublasCall(cublasRnrm2(cublas, size, chebyshev_vec_curr, 1 , &tmp));
      fprintf(stderr,"norm cheby curr: %e ", tmp);
      cublasCall(cublasRnrm2(cublas, size, chebyshev_vec_next, 1 , &tmp));
      fprintf(stderr,"norm cheby next: %e \n", tmp);
      //printVectorDev(brownian_force_ff, 15,  "l. ?467:      ");
      //printVectorDev(chebyshev_vec_last, 15, "cheb467: last ");
      //printVectorDev(chebyshev_vec_curr, 15, "cheb467: curr ");
      //printVectorDev(chebyshev_vec_next, 15, "cheb467: next ");
    }
    // errorcheck of chebyshev polynomial
    assert(isSymmetricDev(mobility_d,lda,size));
    real zMz;
    real alpha = 1, beta = 0;
    cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, mobility_d, lda, brownian_force_ff, 1, &beta,  chebyshev_vec_last, 1));
    cublasCall(cublasRdot(cublas, size, chebyshev_vec_last, 1, brownian_force_ff, 1, &zMz));
    real E_cheby = sqrt(abs(zMz-gaussian_ff_norm*gaussian_ff_norm))/gaussian_ff_norm;
    fprintf(stderr, "The error of the Chebyshev-approximation was %7.3f%% nrm y:%e,  zMz:%e \n",E_cheby*100,gaussian_ff_norm, zMz);
    
    
    
    
  }// end of brownian motion
  cudaCheckError("brownian motion error");
  
  // free everything
  hipFree((void*)resistance_d);
  hipFree((void*)mobility_d);
  hipFree((void*)helper_d);
  cudaCheckError("in mobility");
}

// this calls magma functions to solve the problem: 
// disp * (1+resistance*mobility) = mobility_d *  force_d 
// and returnes disp
// mobility and resistance are square matrizes with size <size> and lda <((size+31)/32)*32>
// force and disp are vectors of size <size>
real sd_iterative_solver(hipblasHandle_t cublas, const real * mobility, const real * resistance, const real * force, int size, real * disp)
{
  int lda = ((size+31)/32)*32;
#ifdef SD_DEBUG
  assert(!hasAnyNanDev(mobility,size*lda));
  assert(!hasAnyNanDev(resistance,size*lda));
  assert(!hasAnyNanDev(force,size));
#endif
  real * mat_a = NULL;
  cuda_safe_mem(hipMalloc( (void**)&mat_a, lda*size*sizeof(real) ));       assert(mat_a != NULL);
  real * mat_a_bak = NULL;
  cuda_safe_mem(hipMalloc( (void**)&mat_a_bak, lda*size*sizeof(real) ));   assert(mat_a_bak != NULL);
  sd_set_zero_matrix<<<192,32>>>(mat_a,size);
  real * mob_force=NULL;
  cuda_safe_mem(hipMalloc( (void**)&mob_force, size*sizeof(real) ));       assert(mob_force !=NULL);
  real * result_checker=NULL;
  cuda_safe_mem(hipMalloc( (void**)&result_checker, size*sizeof(real) ));  assert(result_checker !=NULL);
  // vars for cuBLAS calls
  real alpha=1;
  real beta=0;
  // mat_a = (1+resistance*mobility)
  cublasCall(cublasRgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, size , size ,size, &alpha, mobility, lda,resistance, lda, &beta,mat_a, lda));
  sd_add_identity_matrix<<<128,10>>>(mat_a,size,lda);// TODO: FIXME:  calculate something to set better values ...
  cuda_safe_mem(hipMemcpy(mat_a_bak, mat_a, lda*size*sizeof(real),hipMemcpyDeviceToDevice));
  // mob_force = mobility * force
  cublasCall(cublasRgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mobility, lda, force, 1, &beta, mob_force, 1));
#ifdef SD_DEBUG
  assert(!hasAnyNanDev(mat_a,size*lda));
  assert(!hasAnyNanDev(mob_force,size));
#endif
  int info;
  real res;
  //printVectorDev((real *)force,6,"Kraft");
  //printVectorDev(disp,6,"before");
  info = sd_bicgstab_solver(cublas ,size, mat_a,lda, mob_force, 1e-4, 10*size+100, disp, &res);
  //printVectorDev(disp,6,"after");
  // compary to expected result
  //cuda_safe_mem(hipMemcpy(mat_a, mat_a_bak, lda*size*sizeof(real),hipMemcpyDeviceToDevice));
  
  if (info != 0){
    if (info == 1){
      if (warnings>1) fprintf(stderr, "Iterative solver did not fully converge ... the residuum was %6e\nWe will continue anyway ...\n",res);
    }
    else{ // info == 2 || info == 4
      // try again with reseted displacement vector as initial guess
      sd_set_zero<<<192,16>>>(disp,size);
      info = sd_bicgstab_solver(cublas ,size, mat_a,lda, mob_force, 1e-4, 10*size+100, disp, &res);
      //printVectorDev(disp,6,"after zeroing");
      if (info == 1){
	if (warnings>1) fprintf(stderr, "Iterative solver did not fully converge ... the residuum was %6e\nWe will continue anyway ...\n",res);
      }
      else if (info == 2){
	if(warnings) fprintf(stderr, "Iterative solver failed ... the residuum was %6e\nWe will continue but the results may be problematic ...\n",res);
      }
    }
    // dgetrs is not better - the contrary: results are worse ...
    /*int ipiv[size];
      magma_dgetrf_gpu( size, size,mat_a, lda, ipiv, &info);
      assert(info==0);
      magma_dgetrs_gpu('N', size, 1,
      mat_a, lda, ipiv,
      disp, size, &info);
      assert(info==0);
      // compary to expected result
      cuda_safe_mem(hipMemcpy(mat_a, mat_a_bak, lda*size*sizeof(real),hipMemcpyDeviceToDevice));
      cublasCall(cublasRgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, disp, 1, &beta, result_checker, 1));
      alpha=-1;
      cublasCall(cublasRaxpy( cublas, size, &alpha, mob_force, 1, result_checker, 1));
      alpha=1;
      cublasCall(cublasRdot( cublas, size, result_checker, 1, result_checker, 1,&res));
      if (res > 1e-1){
      fprintf(stderr, "All methods failed :(. The residuum from getrs was %e\n",res);
      //cublasCall(cublasRgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, disp, 1, &beta, result_checker, 1));
      //printVectorDev(mob_force, size, "mob_force");
      //printVectorDev(result_checker, size, "result_checker");
      //printVectorDev(disp, size, "disp");
      //printMatrixDev((real *)mobility,lda,size,"mobility");
      //printMatrixDev((real *)resistance,lda,size,"res");
      //printMatrixDev((real *)mat_a,lda,size,"mat_a");
      }*/
    //magma_int_t magma_dgetrs_gpu( magma_trans_t trans, magma_int_t n, magma_int_t nrhs,
    //				  double *dA, magma_int_t ldda, magma_int_t *ipiv,
    //				  double *dB, magma_int_t lddb, magma_int_t *info);
  }
#ifdef SD_DEBUG
  assert(!hasAnyNanDev(disp,size));
#endif
  //assert(info==0);
  cuda_safe_mem(hipFree((void*)mat_a));
  cuda_safe_mem(hipFree((void*)mat_a_bak));
  cuda_safe_mem(hipFree((void*)mob_force));
  cuda_safe_mem(hipFree((void*)result_checker));
  return res;
}
// this solves iteratively using CG
// disp * (1+resistance*mobility) = mobility_d *  force_d 
// and returnes disp
// mobility and resistance are square matrizes with size <size> and lda <((size+31)/32)*32>
// force and disp are vectors of size <size>
void sd_iterative_solver_cg(hipblasHandle_t cublas, const real * mobility, const real * resistance, const real * force, int size, real * disp)
{
  int lda = ((size+31)/32)*32;
  real * mat_a = NULL;
  cuda_safe_mem(hipMalloc( (void**)&mat_a, lda*size*sizeof(real) ));  assert(mat_a != NULL);
  sd_set_zero_matrix<<<192,32>>>(mat_a,size);
  real * mob_force=NULL;
  cuda_safe_mem(hipMalloc( (void**)&mob_force, size*sizeof(real) ));  assert(mob_force !=NULL);
  real * resid=NULL;
  cuda_safe_mem(hipMalloc( (void**)&resid, size*sizeof(real) ));      assert(resid !=NULL);
  real * p=NULL;
  cuda_safe_mem(hipMalloc( (void**)&p, size*sizeof(real) ));          assert(p !=NULL);
  real * Ap=NULL;
  cuda_safe_mem(hipMalloc( (void**)&Ap, size*sizeof(real) ));         assert(Ap !=NULL);
  real rs_old;
  // count how many iterations we need
  int counter=0;
#ifdef SD_DEBUG
  assert(!hasAnyNanDev(mobility,size*lda));
  assert(!hasAnyNanDev(resistance,size*lda));
#endif
  // vars for cuBLAS calls
  real alpha=1;
  real beta=0;
  // mat_a = (1+resistance*mobility)
  cublasCall(cublasRgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N, size , size ,size, &alpha, mobility, lda,resistance, lda, &beta,mat_a, lda));
  sd_add_identity_matrix<<<128,10>>>(mat_a,size,lda);// TODO: FIXME:  calculate something to set better values ...
  // mob_force = mobility * force
  cublasCall(cublasRgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mobility, lda, force, 1, &beta, mob_force, 1));
  //printMatrixDev(mat_a,lda,size,"A");
  // use mob_force as initial guess
  cublasCall(cublasRcopy(cublas, size,mob_force,1,disp, 1));
  //resid = mob_force-mat_a * disp; //r = b-A*x
  alpha = -1;
  cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, disp, 1, &beta, resid, 1));
  //printVectorDev(resid,size,"-A*disp");
  //printVectorDev(mob_force,size,"solution");
  alpha = 1;
  cublasCall(cublasRaxpy(cublas, size, &alpha, mob_force, 1, resid, 1));
  //printVectorDev(resid,size,"residuum");
  
  // p = resid;                     //p=r
  cublasCall(cublasRcopy(cublas, size,resid,1,p, 1));
  // rsquare_old = r * r;           //rsold=r*r
  cublasCall(cublasRdot( cublas, size, resid, 1, resid, 1, &rs_old));
  std::cerr << counter <<" iterations in integrate_sd::inversion, residuum is "<<rs_old<<std::endl;
  const real req_prec=1e-4;
  if (sqrt(rs_old) < req_prec){
    printf("Converged immediatly\n");
    return;
  }
  while (true){
    // Ap = A * p
    beta = 0;  alpha = 1; cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, mat_a, lda, p, 1, &beta, Ap, 1));
    real pAp;
    cublasCall(cublasRdot( cublas, size, p, 1, Ap, 1, &pAp));
    assert(!isnan(pAp));
    //                              //alpha=rsold/pAp
    alpha = rs_old / pAp;
    // disp += alpha * p            // x=x+alpha * p
    cublasCall(cublasRaxpy(cublas, size, &alpha,  p, 1, disp, 1));
    // resid -= alpha * Ap;         // r=r-alpha * Ap
    real minusalpha=-alpha;
    cublasCall(cublasRaxpy(cublas, size, &minusalpha, Ap, 1, resid, 1));
    real rs_new;
    // rs_new = r * r;              // rsnew = r*r
    cublasCall(cublasRdot( cublas, size, resid, 1, resid, 1, &rs_new));
    if (sqrt(rs_new) < req_prec || counter > 2000){
      break;
    }
    // p=resid+rs_new/rs_old*p      // p = r+rsnew/rsold*p
    alpha = rs_new/rs_old;
    cublasCall(cublasRscal( cublas, size, &alpha, p, 1));
    alpha=1;
    cublasCall(cublasRaxpy( cublas, size, &alpha, resid, 1, p, 1));
    //                              // rsold=rsnew;
    rs_old=rs_new;
    counter++;
    if (counter % 100 == 0){
      std::cerr << counter <<" iterations in integrate_sd::inversion, residuum is "<<rs_new<<std::endl;
    }
  }
  printf("Converged after %d iterations\n",counter);
  cuda_safe_mem(hipFree((void*)mat_a));
  cuda_safe_mem(hipFree((void*)mob_force));
  cuda_safe_mem(hipFree((void*)resid));
  cuda_safe_mem(hipFree((void*)p));
  cuda_safe_mem(hipFree((void*)Ap));
}

// BICGSTAB-Solver
// implimented as given in Numerik linearer Gleichungssysteme by Prof. Dr. Andreas Meister
// this solves A*x=b
// cublas a handle for cublas
// size   the size n of the matrix
// A      the given n*n matrix (in)
// lda    the leading demension of A
// b      the given solution vector (in)
// tol    requested tolerance of the solution
// maxit  maximum number of iterations
// x      the requested solution with an initial guess (in/out)
// returns 0 on success, else error code
int sd_bicgstab_solver(hipblasHandle_t cublas ,int size, real * A,int lda, real * b, real tol, int maxit, real * x, real * res){
  // vector malloc
  real * r0=NULL;
  cuda_safe_mem(hipMalloc( (void**)&r0, size*sizeof(real) ));       assert(r0 != NULL);
  real * r=NULL;
  cuda_safe_mem(hipMalloc( (void**)&r, size*sizeof(real) ));        assert(r != NULL);
  real * p=NULL;
  cuda_safe_mem(hipMalloc( (void**)&p, size*sizeof(real) ));        assert(p != NULL);
  real * v=NULL;
  cuda_safe_mem(hipMalloc( (void**)&v, size*sizeof(real) ));        assert(v != NULL);
  real * t=NULL;
  cuda_safe_mem(hipMalloc( (void**)&t, size*sizeof(real) ));        assert(t != NULL);
  real * test=NULL;
  cuda_safe_mem(hipMalloc( (void**)&test, size*sizeof(real) ));     assert(test != NULL);
  // constants
  real eps;
  if (sizeof(real) == sizeof(double)){
    eps = 1e-15;
  } else {
    eps = 1e-7;
  }
  eps = min(eps,tol*1e-2);
  // other variables
  real alpha=1;
  real beta=0;
  real tolb;
  // compute the norm of b
  real normb;
  cublasCall(cublasRdot( cublas, size, b, 1, b, 1, &normb));
  normb=sqrt(normb);
  //tolb=min(tol*size, tol*normb); // tol is not realy usefull as this wont be reached ... at least without preconditioning
  //tolb=max(normb*eps, tolb);
  tolb=tol*normb;
  // r0 = b-A*x
  alpha = -1;
  cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, x, 1, &beta, r0, 1));
  alpha = 1;
  cublasCall(cublasRaxpy(cublas, size, &alpha, b, 1, r0, 1));
  // r = r0
  cublasCall(cublasRcopy(cublas, size,r0,1,r, 1));
  // rr0 = r*r0
  real rr0;
  cublasCall(cublasRdot( cublas, size, r0, 1, r0, 1, &rr0));
  // p =r
  cublasCall(cublasRcopy(cublas, size,r0,1,p, 1));
  // normr=norm(r)
  real normr=sqrt(rr0);
  int iteration=0;
  real lastnorm=normr;
  real initnorm=normr;
  // check for conversion or max iterations
  while (iteration < maxit && normr >= tolb){
    // v=A*p
    cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, p, 1, &beta, v, 1));
    // vr0 = v*r0
    real vr0;
    cublasCall(cublasRdot( cublas, size, v, 1, r0, 1, &vr0));
    if (fabs(vr0) < eps || rr0 == 0){
      if (fabs(vr0) < eps){
	if (warnings > 1) fprintf(stderr, "BICGSTAB break-down.\n");
      }else{
	if (warnings > 1) fprintf(stderr, "BICGSTAB solution stagnates.\n");
      }
      cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
      res[0] = normr;
      if (tolb > normr)     { return 0;}
      if (tolb*100 > normr) { return 1;}
      else                  { return 2;}
    }
    // alpha = rr0/vr0
    real myAlpha=rr0/vr0;
    real minusMyAlpha = -myAlpha;
    // s = r - alpha v
    //cublasCall(cublasRcopy(cublas, size,r,1,s, 1));
    cublasCall(cublasRaxpy(cublas, size, &minusMyAlpha, v, 1, r, 1)); //s->r
    // t = A * s
    cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, r, 1, &beta, t, 1));// s->r
    // ts = s * t
    real ts;
    cublasCall(cublasRdot( cublas, size, t, 1, r, 1, &ts));// s->r
    // tt = t * t
    real tt;
    cublasCall(cublasRdot( cublas, size, t, 1, t, 1, &tt));
    if (abs(tt)<eps || ts == 0){
      fprintf(stderr,"Exit: abs(tt)<eps || ts == 0\n");
      if (warnings > 1) fprintf(stderr, "BICGSTAB break-down.\n");
      cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
      res[0] = normr;
      if (tolb > normr)     { 
	//fprintf(stderr, "0: tolb: %e normr: %e \n",tolb, normr);
	return 0;}
      if (tolb*100 > normr) { 
	//fprintf(stderr, "1: tolb: %e normr: %e \n",tolb, normr);
	return 1;}
      else                  { 
	//fprintf(stderr, "2: tolb: %e normr: %e \n",tolb, normr);
	return 2;}
    }
    // omega = ts/tt
    real myOmega=ts/tt;
    // x = x + alpha p + omega s
    cublasCall(cublasRaxpy(cublas, size, &myAlpha, p, 1, x, 1));
    cublasCall(cublasRaxpy(cublas, size, &myOmega, r, 1, x, 1));
    // copyback of s to r
    // r = s - omega t
    real minusMyOmega=-1*myOmega;
    cublasCall(cublasRaxpy(cublas, size, &minusMyOmega, t, 1, r, 1));
    //myOmega*=-1;
    // r1r0 = r * r0
    real r1r0;
    cublasCall(cublasRdot( cublas, size, r, 1, r0, 1, &r1r0));
    // beta = (alpha * r1r0 ) / (omega rr0)
    real myBeta = (myAlpha*r1r0)/(myOmega*rr0);
    if (abs(myBeta)>1/eps){
      fprintf(stderr,"Exit: abs(myBeta)<1/eps\n");
      cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
      res[0] = normr;
      if (tolb > normr)     { return 0;}
      if (tolb*100 > normr) { return 1;}
      else                  { return 2;}
    }
    // p = r + beta ( p - omega v)= beta p + r - beta omega v
    cublasCall(cublasRscal(cublas, size, &myBeta, p, 1));
    cublasCall(cublasRaxpy(cublas, size, &alpha, r, 1, p, 1));
    alpha=-myBeta*myOmega;
    cublasCall(cublasRaxpy(cublas, size, &alpha, v, 1, p, 1));
    alpha=1;
    rr0=r1r0;
    real r1r1;
    cublasCall(cublasRdot( cublas, size, r, 1, r, 1, &r1r1));
    normr=sqrt(r1r1);
    iteration++;
    if (lastnorm*sqrt(eps) > normr){ // restart
      //fprintf(stderr, "recalculation r\n");
      cublasCall(cublasRcopy(cublas, size,b,1,r, 1));
      alpha=-1;beta=1;
      cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, x, 1, &beta, r, 1));
      alpha= 1;beta=0;
      cublasCall(cublasRdot( cublas, size, r, 1, r, 1, &rr0));
      normr=sqrt(rr0);
      lastnorm = normr;
      // r = r0
      cublasCall(cublasRcopy(cublas, size,r,1,r0, 1));
      // p =r
      cublasCall(cublasRcopy(cublas, size,r,1,p, 1));
    }
    if (iteration%500000 == 0){ // enable debugging by setting this to a lower value
      real realnorm;
      {// recalculate normr
	cublasCall(cublasRcopy(cublas, size,b,1,test, 1));
	alpha=-1;beta=1;
	cublasCall(cublasRgemv(cublas, HIPBLAS_OP_T, size, size, &alpha, A, lda, x, 1, &beta, test, 1));
	alpha= 1;beta=0;
	cublasCall(cublasRdot( cublas, size, test, 1, test, 1, &realnorm));
	realnorm=sqrt(realnorm);
      }
      fprintf(stderr,"  Iteration: %6d Residuum: %12f RealResiduum: %12f\n",iteration, normr, realnorm);
    }
    if (initnorm*1e10 < normr){ // somehow our solution explodes ...
      if (warnings) fprintf(stderr, "BICGSTAB did not converge, residuum exploded. Aborting.\n");
      cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
      res[0]= normr;
      return 4;
    }
  }
  res[0]=normr;
  if (normr > tolb*1.01){
    fprintf(stderr, "BICGSTAB solution did not converge after %d iterations. Error was %e1 %% to high.\n",iteration,(normr/tolb-1)*100);
    cuda_safe_mem(hipFree((void*)r0));cuda_safe_mem(hipFree((void*)r));cuda_safe_mem(hipFree((void*)p));cuda_safe_mem(hipFree((void*)v));cuda_safe_mem(hipFree((void*)t));cuda_safe_mem(hipFree((void*)test));
    if (tolb*100 > normr){
      fprintf(stderr, "1: tolb: %e normr: %e \n",tolb, normr);
      return 1;
    } else {
      fprintf(stderr, "2: tolb: %e normr: %e \n",tolb, normr);
      return 2;
    }
  }
  //fprintf(stderr, "BICGSTAB solution did converge after %d iterations.\n",iteration);
  
  cuda_safe_mem(hipFree((void*)r0));
  cuda_safe_mem(hipFree((void*)r));
  cuda_safe_mem(hipFree((void*)p));
  cuda_safe_mem(hipFree((void*)v));
  cuda_safe_mem(hipFree((void*)t));
  cuda_safe_mem(hipFree((void*)test));
  return 0;
}

// calculates the largest and snalles eigenvalue of the matrix
// size        : size of the eigenvector / the matrix           (IN)
// mobility_d  : handle of the mobility matrix (on the device)  (IN)
// lambda_min  : smalles eigenvalue                            (OUT)
// lambda_max  : largest eigenvalue                            (OUT)
void calculate_maxmin_eigenvalues(int size,real *mobility_d,real * lambda_min,real * lambda_max){
  int lda = ((size+31)/32)*32;
  int maxit=max(500,size);
  int IDO;
  char BMAT='I'; // standard eigenvalue problem
  char WHICH[]="SR"; // start with largest eigenvalue
  int NEV = 1; // only one eigenvalue
  // TUNING: these could be adjusted?
  real TOL=1e-1;
  if (sizeof(double) == sizeof(real)){
    TOL=max(1e-12,TOL);
  } else {
    TOL=max(1e-6,TOL);
  }
  // TUNING: make some tests to find good value ...
  int NCV=min(size, 6); // must be at least 3, but a bit bigger should be better ...
  int LDV=lda;
  int LWORKL=3*NCV*(NCV + 2);
  int mode=1;
  int IPARAM[11] = {1,0,maxit,1,0,0,mode,0,0,0,0};
  int IPNTR[14];
  int INFO=0;
  real RESID[size];
  real V[LDV*NCV];
  real WORKD[3*size];
  real WORKL[LWORKL];
  real * vec_in_d;
  real * vec_out_d;
  cuda_safe_mem(hipMalloc((void**)&vec_in_d , lda*sizeof(real)));
  cuda_safe_mem(hipMalloc((void**)&vec_out_d, lda*sizeof(real)));
  for (int minmax=0;minmax<2;minmax++){
    IDO=0;
    if (minmax){
      sprintf(WHICH,"LR");
      INFO=1;
      IPARAM[2]=maxit;
      TOL=1e-3;
    }
    while (IDO != 99){
      //dnaupd_(&IDO,&BMAT,&N,WHICH,&NEV,&TOL,RESID.memptr(),&NCV,V.memptr(),&LDV,IPARAM,IPNTR,WORKD,WORKL,&LWORKL,&INFONAUP);
      rnaupd(&IDO,&BMAT,&size, WHICH, &NEV, &TOL, RESID, &NCV, V, &LDV, IPARAM, IPNTR, WORKD, WORKL, &LWORKL, &INFO);
      switch (IDO){
      case 1:
	cuda_safe_mem(hipMemcpy(vec_in_d,WORKD+IPNTR[0]-1,size*sizeof(real),hipMemcpyHostToDevice));
	{
	  real alpha=1, beta=0;
	  cublasCall(cublasRgemv( cublas, HIPBLAS_OP_T, size, size, &alpha, mobility_d, lda, vec_in_d, 1, &beta, vec_out_d, 1));
	}
	cuda_safe_mem(hipMemcpy(WORKD+IPNTR[1]-1,vec_out_d,size*sizeof(real),hipMemcpyDeviceToHost));
	break;
      case -1:
      case 2:
      case 3:
      case 4:
	fprintf(stderr,"Error in %s l. %d: unexpected work from rnaupd: %d: Not Implemented!\n",__FILE__,__LINE__,IDO);
	break;
      case 99: //we are done
	break;
      default:
	fprintf(stderr,"Error in %s l. %d: unexpected work from rnaupd: %d: Not Understood!\n",__FILE__,__LINE__,IDO);
	break;
      }
    } 
    fprintf(stderr,"calculationg eigenvalue needed %d iterations and %d gemv operations (tolerance is %e, EW is %e).\n"
	    ,IPARAM[2], IPARAM[8], TOL,WORKL[IPNTR[5]-1]);
    if (INFO){
      fprintf(stderr,"Unexpected return value in %s l. %d from rnaupd_: %d\n",__FILE__,__LINE__,INFO);
    }
    if (WORKL[IPNTR[5]-1]<0 && TOL > 1e-3){
      minmax--;
      TOL=1e-4;
      INFO=1;
      IPARAM[2]=maxit*100;
    }
    if (minmax){ // make them a bit larger/smaller to be sure that we are in the interval of interrest ...
      *lambda_max=WORKL[IPNTR[5]-1]*(1+TOL);
    } else {
      *lambda_min=WORKL[IPNTR[5]-1]*(1-TOL);
    }
  }
  /* FORTRAN Comments ...
     c          IPNTR(6): pointer to the real part of the ritz value array     
     c                    RITZR in WORKL.                                          
     c          IPNTR(7): pointer to the imaginary part of the ritz value array    
     c                    RITZI in WORKL.                                          
     c          IPNTR(8): pointer to the Ritz estimates in array WORKL associated
     c                    with the Ritz values located in RITZR and RITZI in WORK
  */
}

// lambda_min   : the lower boundery
// lambda_max   : the upper boundery of the interval
// tol          : the given tollerance which should be achieved
// coefficents  : the pointer where the data will be stored
real calculate_chebyshev_coefficents(real lambda_min, real lambda_max, real tol,real ** coefficents){
  // use chebyshev-gausquadrature: https://en.wikipedia.org/wiki/Chebyshev%E2%80%93Gauss_quadrature
  const int steps=1024*128; // with 1024 it should fit in L1
  unsigned int N=1024; // guess the number of coefficents we need, if more are needed -> realocate
  if (*coefficents==NULL){
    *coefficents = (real *)malloc(N*sizeof(real));
  } else {
    *coefficents = (real *)realloc(*coefficents,N*sizeof(real));
  }
  real * current_polynome=NULL;
  real * last_polynome=NULL;
  last_polynome    = (real *) malloc(steps * sizeof(real));
  current_polynome = (real *) malloc(steps * sizeof(real));
  real x[steps];
  real weight_and_func[steps];
  real lambda_m=lambda_max-lambda_min;
  real lambda_p=lambda_max+lambda_min;
  //fprintf(stderr,"lambda_min: %e lambda_max: %e  ", lambda_min, lambda_max);
  //fprintf(stderr,"lambda_minus: %e lambda_plusminus: %e\n", lambda_m, lambda_pm);
  // init
  real fac=2./steps;
  //fprintf(stderr,"fac: %e\n", fac);
  double ai=0;
  double a1=0;
  for (int i=0;i<steps;i++){
    last_polynome[i]=1; //T0(x)=1
    //real x = -1.+(i*2.+1.)/steps;
    x[i]=cos((2.*i+1)/2./steps*M_PI);
    current_polynome[i]= x[i];  //T1(x)=x
    weight_and_func[i]=fac*1./sqrt(x[i]*lambda_m/2.+lambda_p/2.);// /sqrt(1-x*x);// this could be big, but should not be inf
    ai+=weight_and_func[i];
    a1+=weight_and_func[i]*x[i];
    if (i%100 == 0){
      ;//fprintf(stderr,"i: %d a0: %e  a1: %e \n",i,ai,a1);
    }
  }
  real error;
  int loop=0;
  //fprintf(stderr,"%s l. %d: a[%d]: %e\n",__FILE__,__LINE__,0,ai);
  //fprintf(stderr,"%s l. %d: a[%d]: %e\n",__FILE__,__LINE__,1,a1);
  (*coefficents)[loop]=ai/2.;
  loop++;
  (*coefficents)[loop]=a1;
  //double sumfacmax=lambda_max;
  //double totalsum=abs(ai)+abs(a1)*sumfacmax;
  double totalsum=abs(ai)+abs(a1);
  //sumfacmax*=lambda_max;
  const int miniloop=10;
  do{
    error=0;
    do {
      std::swap(last_polynome,current_polynome);
      //{	real * tmp       = last_polynome; last_polynome    = current_polynome; current_polynome = tmp; }
      //printf("addresses: 0x%08x  0x%08x ",last_polynome,current_polynome);
      ai=0;
      for (int i=0;i<steps;i++){
	//real x = -1.+(i*2.+1.)/steps;
	current_polynome[i]=-1.*current_polynome[i]+((real)2)*x[i]*last_polynome[i];
	//printf ("%e %e %e\n", x, weight_and_func[i], current_polynome[i]);
	ai+=current_polynome[i]*weight_and_func[i];
      }
      //fprintf(stderr,"%s l. %d: a[%d]: %e\n",__FILE__,__LINE__,loop,ai);
      //printf("\n\n");
      //sumfacmax*=lambda_max;
      error+=abs(ai);//*sumfacmax;
      totalsum+=abs(ai);//*sumfacmax;
      (*coefficents)[loop]=ai;
      loop++;
    } while (loop%miniloop);
    if (loop+miniloop > N){
      N*=2;
      *coefficents=(real *)realloc(*coefficents,N*sizeof(real));
    }
  } while ((error > tol*totalsum || loop < 20 ) && loop < sqrt(steps));
  if (loop >=steps/10 -1 ){
    fprintf(stderr,"to few steps to get sufficent results in %s l. %d\n",__FILE__,__LINE__);
  }
  error=0;
  while (error < tol*totalsum){ // approximate error
    loop--;
    error+=abs((*coefficents)[loop]);//*sumfacmax;
    //sumfacmax/=lambda_max;
  }
  fprintf(stderr,"sum: %e   error: %e",totalsum,error);
  loop++;
  free(last_polynome);
  free(current_polynome);
  fprintf(stderr,"loops: %d\n",loop);
  return loop;
}


// check whether there was any cuda error so far.
// do not use this function directly but use the macro cudaCheckError(const char *msg);
// which requires only the first paramter
// PARAMTERS:
// msg   : the message which should be printed in case of an error
// file  : the file in which the function is called
// line  : the line in which the function is called
void _cudaCheckError(const char *msg, const char * file, const int line)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err)
    {
      std::cerr <<  "Cuda error:" <<  msg << ": '" <<  hipGetErrorString( err) << "' in "<<file << " l. "<<line<<"\n";
      errexit();
    }
}


/* *************************************************************************************************************** *
 * ********************************************      CUDA-KERNELS     ******************************************** *
 * *************************************************************************************************************** */


// This computes the farfield contribution of the mobility
// r is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N is the number of particles
// self_mobility is 1./(6.*PI*eta*a)
// a is the particle radius
// mobility is the mobility matrix which will be retruned
// L_d is the boxlength
#define mydebug(str,...)
// if (threadIdx.x < 3 && (blockIdx.x == 0 || blockIdx.x == 1)){printf("line: %d thread: %2d, block: %2d "str,__LINE__,threadIdx.x,blockIdx.x,__VA_ARGS__);}
__global__ void sd_compute_mobility_matrix(real * r, int N, real self_mobility, real a, real * L_g, real * mobility){
  real mypos[3];
  const int lda=((3*N+31)/32)*32;
  __shared__ real L[3];
  __shared__ real cachedPos[3*numThreadsPerBlock];
  __shared__ real writeCache[3*numThreadsPerBlock];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < 3){ // copy L to shared memory
    //mydebug("0x%08x  \n",L_g + threadIdx.x);
    L[threadIdx.x]=L_g[threadIdx.x];
  }
  __syncthreads();
  // get data for myposition - using coalscaled memory access
  for (int l=0;l<3;l++){
    mydebug(" 0x%08x -> 0x%08x  \n",numThreadsPerBlock*(l+blockIdx.x*3)+threadIdx.x,numThreadsPerBlock*l+threadIdx.x);
    cachedPos[numThreadsPerBlock*l+threadIdx.x] = r[numThreadsPerBlock*(l+blockIdx.x*3)+threadIdx.x];
  }
  __syncthreads();
  for (int l=0;l<3;l++){
    mypos[l] = cachedPos[threadIdx.x*3+l];
    mydebug("mypos[%d]:  %e\n",l,mypos[l]);
  }

  /*if (i < N){
    // first write the self contribution
#pragma unroll
    for (int k=0; k < DIM; k++){
      //#pragma unroll
      //for (int l=0; l < DIM; l++){
      //mobility[myindex(DIM*i+k,DIM*i+l)]=0;
      //}
      mobility[myindex(DIM*i+k,DIM*i+k)]=self_mobility;
    }
    }*/
  for (int offset=0;offset<N;offset+=numThreadsPerBlock){
    mydebug("offset: %d\n",offset)
    // copy positions to shared memory
#pragma unroll
    for (int l=0;l<3;l++){
      mydebug("fuu:: 0x%08x  0x%08x  0x%08x  0x%08x %e\n",r, offset*3,numThreadsPerBlock*l,threadIdx.x,cachedPos[numThreadsPerBlock*l+threadIdx.x]);
      cachedPos[numThreadsPerBlock*l+threadIdx.x] = r[offset*3+numThreadsPerBlock*l+threadIdx.x];
    }
    __syncthreads();
    if (i < N){
      for (int j=offset;j<min(offset+numThreadsPerBlock,N);j++){
	real dr[DIM];
	real dr2=0;
#pragma unroll 3
	for (int k=0;k<DIM;k++){
	  dr[k]=mypos[k]-cachedPos[DIM*(j-offset)+k]; // r_ij
	  dr[k]-=rint(dr[k]/L[k])*L[k]; // fold back
	  dr2+=dr[k]*dr[k];
	}
	dr2=max(dr2,0.01);
	real drn= sqrt(dr2); // length of dr
	real b = a/drn;
      
	/*if (0.5 < b){  // drn < 2*a
	  /*real t=3./32./drn/a*self_mobility;
	  real t2=(1-9./32.*drn/a)*self_mobility;
	  for (k=0; k < DIM; k++){
	  for (l=0;l < DIM; l++){
	  mobility[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*t;
	  }
	  mobility[myindex(DIM*i+k,DIM*j+k)]+=t2;
	  }*/ // this should not happen ...
	// python implementation:
	//T=one*(1-9./32.*drn/a)+3./32.*dr*drt/drn/a;
	//}
	real t,t2;
	// this also catches the case i == j
	if (0.5 < b){  // drn < 2*a
	  t=0;
	  t2=0;
	  if (i==j){
	    t2=self_mobility;
	  }
	} else {
	  real b2=(a*a)/dr2;
	  // Rotne Prager
	  //t=(0.75-1.5*b2)*b/dr2*self_mobility;
	  //t2=(0.75+0.5*b2)*b*self_mobility;
#warning "Wrong Mobility in Farfield - to assure positive definiteness"
	  // the /5. is to much ... but like this it seems to work
	  t=(0.75-1.5*b2)*b/dr2*self_mobility/5.;
	  t2=(0.75+0.5*b2)*b*self_mobility/5.;
	}
	//mobility[threadIdx.x]=3+threadIdx.x;
	real tmp_el13;
#pragma unroll 3
	for (int k=0; k < DIM; k++){
	  if (k ==0){ // these ifs should be removed at compile time ... after unrolling
#pragma unroll 3
	    for (int l=0;l < 3; l++){
	      //mobility[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*t;
	      writeCache[3*threadIdx.x+l]=dr[k]*dr[l]*t;
	    }
	  }
	  else if(k==1){
	    tmp_el13 = writeCache[3*threadIdx.x+2];
	    writeCache[3*threadIdx.x+0]=writeCache[3*threadIdx.x+1];
#pragma unroll 2
	    for (int l=1;l < DIM; l++){
	      //mobility[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*t;
	      writeCache[3*threadIdx.x+l]=dr[k]*dr[l]*t;
	    }	
	  }
	  else{
	    writeCache[3*threadIdx.x+0]=tmp_el13;
	    writeCache[3*threadIdx.x+1]=writeCache[3*threadIdx.x+2];
	    writeCache[3*threadIdx.x+2]=dr[k]*dr[2]*t;
	  }
	  writeCache[3*threadIdx.x+k]+=t2;
	    
	  __syncthreads();
	  //int max = min(blockDim.x, N-(blockIdx.x*blockDim.x));
	  int max = min(blockDim.x,N-blockDim.x*blockIdx.x);
	  for (int l=0;l<3;l++){
	    //mobility[(DIM*j+k)*3*N+blockIdx.x*blockDim.x+threadIdx.x+blockDim.x*l]=writeCache[threadIdx.x+blockDim.x*l];
	    mobility[(DIM*j+k)*lda+blockIdx.x*blockDim.x*3+max*l+threadIdx.x]=writeCache[max*l+threadIdx.x];
	  }
	  //mobility[myindex(DIM*i+k,DIM*j+k)]+=t2;
	}
	// python implementation:
	// T=one*(0.75+0.5*b2)*b+(0.75-1.5*b2)*b*drt*dr/dr2;
	//} // if (j <N)
      } // for (j = ...
    } // if (i < N)
  }// for offset = ...
}

#undef mydebug
#define mydebug(str,...)
// if (threadIdx.x < 3 && blockIdx.x < 2){printf("line: %d thread: %2d, block: %2d "str,__LINE__,threadIdx.x,blockIdx.x,__VA_ARGS__);}
// this computes the near field as a  ResistanceMatrix
// r             : is the vector of [x_1, y_1, z_1, x_2, y_2, z_2, ...]
// N             : is the number of particles
// self_mobility : is 1./(6.*PI*eta*a)
// a             : is the particle radius
// L_d           : is the boxlength
// resistance    : is the resistance matrix which will be retruned
// myInfo        : contains infos about the operation:
//                myInfo[0] : number of overlapping particles
//                myInfo[1] : number of interacting particles (via nf)
//                myInfo[2] : max number of interacting particles per particle
__global__ void sd_compute_resistance_matrix(real * pos, int N, real self_mobility, real a, real * L_g, real * resistance, int * myInfo){
  //__shared__ real myPos[3*numThreadsPerBlock];
  int interactions=0;
  real mypos[3];
  __shared__ real L[3];
#ifdef SD_USE_FLOAT
  __shared__ real cachedPos[4*numThreadsPerBlock];
#else
  __shared__ real cachedPos[3*numThreadsPerBlock];
#endif
  const int lda=(((N*3)+31)/32)*32;
  //__shared__ real myresistance[6*numThreadsPerBlock];
  real myresistance[6]={0,0,0,0,0,0};
  //__shared__ real otherresistance[6*numThreadsPerBlock];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < 3){ // copy L to shared memory
    L[threadIdx.x]=L_g[threadIdx.x];
  }
  //__syncthreads();
  // get data for myposition - but coalscaled
  /*for (int l=0;l<3;l++){
    myPos[threadIdx.x+l*numThreadsPerBlock] = r[threadIdx.x+l*numThreadsPerBlock+blockIdx.x*blockDim.x*3];
    }*/
  for (int l=0;l<3;l++){
    //mydebug("pos: 0x%010x   offset: 0x%010x\n",pos,threadIdx.x+l*numThreadsPerBlock+blockIdx.x*blockDim.x*3);
    cachedPos[threadIdx.x+l*numThreadsPerBlock] = pos[threadIdx.x+l*numThreadsPerBlock+blockIdx.x*blockDim.x*3];
  }

  __syncthreads();
  for (int d=0;d<3;d++){
    mypos[d] = cachedPos[threadIdx.x*3+d];
  }
  
  //for (int i = idx; i < N; i+=blockDim.x*gridDim.x){
  /*if (i < N){
#pragma unroll 3
    for (int k=0; k < DIM; k++){
#pragma unroll 3
      for (int l=0;l < DIM; l++){
	resistance[myindex(DIM*i+k,DIM*i+l)]=0; // we will add some terms on the diagonal, so set it to zero before
      }
    }
  }*/
  for (int offset=0;offset<N;offset+=numThreadsPerBlock){
    // copy positions to shared memory
#pragma unroll
    for (int l=0;l<3;l++){
      cachedPos[threadIdx.x+l*numThreadsPerBlock] = pos[threadIdx.x+l*numThreadsPerBlock+offset*3];
    }
    __syncthreads();
    for (int j=offset;j<min(offset+numThreadsPerBlock,N);j++){
      real dr[DIM];
      real dr2=0;
#pragma unroll
      for (int k=0;k<DIM;k++){
	dr[k]=mypos[k]-cachedPos[3*(j-offset)+k]; // r_ij
	dr[k]-=L[k]*rint(dr[k]/L[k]); // fold back
	dr2+=dr[k]*dr[k];
	mydebug("dr[%d]: %f\n",k,dr[k]);
      }
#ifdef SD_RESISTANCE_CORRECT
      mydebug("dr2: %f\n",dr2);
      real r2bcorr_diag_self     = 0;
      real r2bcorr_diag_mix      = 0;
      real r2bcorr_offdiag_self  = 0;
      real r2bcorr_offdiag_mix   = 0;
#else
      real offdiag_fac=0;
      real diag_fac=0;
#endif
      if (i >= N || i ==j || j >= N){
	;
      }
      else if (dr2 < 4*a*4*a){
	if (!(2*a*2*a < dr2 )){
	  atomicAdd(myInfo,1); // count overlapping particles
	}
	else {// 2*a < drn < 4*a 
	  interactions++;
	  // python code:
	  // # Use only singular therms, namely to order O(s_ij^0)                                                                  
	  // T=(1./4./s-1/4-9./40.*ls)*dr*drt/dr2
	  // #           ^ this additonal constant is so that the mobility is smooth
	  // # c.f. N.-Q. Nguyen and A. J. C. Ladd, PHYSICAL REVIEW E 66, 046708 (2002) equation (34)                               
	  // T+=1./6.*ls*(-one+dr*drt/dr2)
	  // R[3*i:3*i+3,3*j:3*j+3]=-T
	  // R[3*i:3*i+3,3*i:3*i+3]+=T
	  real drn= sqrt(dr2); // length of dr
	  real s = drn/a-2;
	  real ls = log(s);
	  
	  mydebug("ls: %e \n",ls);
#ifdef SD_RESISTANCE_CORRECT
	  real const t_c=-0.125+9./40.*log(2.)+3./112.*2.*log(2.);
	  real offdiag_fac =(-0.25/s+9./40.*ls+3./112.*s*ls-t_c)/dr2;
	  real diag_fac    =(1./6.*ls);
#else
	  real const t_c=-0.125+9./40.*log(2.)+3./112.*2.*log(2.);
	  real const t2_c=2./6.*log(2.);
	  offdiag_fac =(-0.25/s+9./40.*ls+3./112.*s*ls-t_c)/dr2/self_mobility;
	  diag_fac    =(1./6.*ls-t2_c)/self_mobility;
#endif
#ifdef SD_RESISTANCE_CORRECT
	  real dr4=dr2*dr2;
	  real dr6=dr4*dr2;
	  // constants for correction
	  const real dr_c1 = 4;
	  const real dr_c2 = 4*4;
	  const real dr_c3 = 4*4*4;
	  const real dr_c4 = 4*4*4*4;
	  const real dr_c5 = 4*4*4*4*4;
	  const real dr_c6 = 4*4*4*4*4*4;
	  const real r2bcorr_diag_self_c    = (4.*dr_c6)/(4.*dr_c6-9.*dr_c4+12.*dr_c2-4.)         ;
	  const real r2bcorr_diag_mix_c     = (9.*dr_c5-4.*dr_c3)/(4.*dr_c6-9.*dr_c4+12.*dr_c2-4.);
	  const real r2bcorr_offdiag_self_c = 16.*dr_c2 /(16.*dr_c2-25)                            - 2./6.*log(2.);
	  const real r2bcorr_offdiag_mix_c  = 20.*dr_c1 /(16.*dr_c2-25)                            - 2./6.*log(2.);
	  // real computation
	  r2bcorr_diag_self     = diag_fac    - 1./(1-9./4./dr2+3./dr4-1./dr6)                     + r2bcorr_diag_self_c;
	  r2bcorr_diag_mix      = diag_fac    - (6.*dr4*drn-4.*dr2*drn)/(4.*dr6-9.*dr4+12.*dr2-4.) + r2bcorr_diag_mix_c;
	  r2bcorr_offdiag_self  = offdiag_fac - 1./(1.-25./16./dr2)                                + r2bcorr_offdiag_self_c;
	  r2bcorr_offdiag_mix   = offdiag_fac - 1./(16./20.*drn-25./20./drn)                       + r2bcorr_offdiag_mix_c;
	  r2bcorr_diag_self    /= self_mobility;
	  r2bcorr_diag_mix     /= self_mobility;
	  r2bcorr_offdiag_self /= self_mobility;
	  r2bcorr_offdiag_mix  /= self_mobility;
#endif
	}
      }
      if (i < N){
#pragma unroll 3
	for (int k=0; k < DIM; k++){
#pragma unroll 3
	  for (int l=0;l < DIM; l++){
#ifdef SD_RESISTANCE_CORRECT
	    resistance[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*r2bcorr_offdiag_mix;
#else
	    resistance[myindex(DIM*i+k,DIM*j+l)]=dr[k]*dr[l]*offdiag_fac;
#endif
	    
	    //resistance[myindex(DIM*i+k,DIM*i+l)]-=dr[k]*dr[l]*t;
	  }
#ifdef SD_RESISTANCE_CORRECT
	  myresistance[k]-=dr[k]*dr[k]*r2bcorr_offdiag_self;
	  resistance[myindex(DIM*i+k,DIM*j+k)]+=r2bcorr_diag_mix;
	  myresistance[k]-=r2bcorr_diag_self;
#else
	  myresistance[k]-=dr[k]*dr[k]*offdiag_fac;
	  resistance[myindex(DIM*i+k,DIM*j+k)]+=diag_fac;
	  myresistance[k]-=diag_fac;
	  mydebug(""
#endif
	}
      }
#ifdef SD_RESISTANCE_CORRECT
      myresistance[3]-=r2bcorr_offdiag_self*dr[0]*dr[1];
      myresistance[4]-=r2bcorr_offdiag_self*dr[0]*dr[2];
      myresistance[5]-=r2bcorr_offdiag_self*dr[1]*dr[2];
#else
      myresistance[3]-=offdiag_fac*dr[0]*dr[1];
      myresistance[4]-=offdiag_fac*dr[0]*dr[2];
      myresistance[5]-=offdiag_fac*dr[1]*dr[2];
#endif
      // python implementation:
      //T=one*(1-9./32.*drn/a)+3./32.*dr*drt/drn/a;
    }
    
    /*else{ // set the block to zero
    // it might be faster to set everything in the beginning to zero ...
    // or use sparse matrices ...
#pragma unroll 3
	  for (int k=0; k < DIM; k++){
#pragma unroll 3
	    for (int l=0;l < DIM; l++){
	      resistance[myindex(DIM*i+k,DIM*j+l)]=0;
	    }
	    }  
	  }*/
    
  }
  if ( i < N){
#pragma unroll
    for (int k=0;k<3;k++){
      resistance[myindex(DIM*i+k,DIM*i+k)]=myresistance[k];
    }
    resistance[myindex(DIM*i+0,DIM*i+1)]=myresistance[3];
    resistance[myindex(DIM*i+1,DIM*i+0)]=myresistance[3];
    resistance[myindex(DIM*i+0,DIM*i+2)]=myresistance[4];
    resistance[myindex(DIM*i+2,DIM*i+0)]=myresistance[4];
    resistance[myindex(DIM*i+1,DIM*i+2)]=myresistance[5];
    resistance[myindex(DIM*i+2,DIM*i+1)]=myresistance[5];
  }
  __syncthreads();
  int * sharedInteractions = (int *) cachedPos; // reuse shared memory
  int * maxInteractions    = sharedInteractions + blockDim.x*2;
  sharedInteractions[threadIdx.x]=interactions;
  sharedInteractions[threadIdx.x+blockDim.x]=0;
  maxInteractions[threadIdx.x]   =interactions;
  maxInteractions[threadIdx.x+blockDim.x]   =0;
  for (int t=(blockDim.x+1)/2;t>1;t=(t+1)/2){
    if (threadIdx.x < t){
      sharedInteractions[threadIdx.x]+=sharedInteractions[threadIdx.x+t];
      sharedInteractions[threadIdx.x+t]=0;
      maxInteractions[threadIdx.x]=max(maxInteractions[threadIdx.x+t],maxInteractions[threadIdx.x]);
    }
    __syncthreads();
  }
  if (threadIdx.x==0){
    sharedInteractions[0]+=sharedInteractions[1];
    atomicAdd(myInfo+1, sharedInteractions[0]);
    maxInteractions[0]=max(maxInteractions[0],maxInteractions[1]);
    atomicMax(myInfo+2, maxInteractions[0]);
  }
}

__global__ void sd_compute_brownian_force_nearfield(real * r,real * gaussian,int N,real * L_g, real a, real self_mobility,real * brownian_force_nf){
  const int gaussian_ldd=((N+31)/32)*32;
  int interactions=0;
  real mypos[3];
  real writeCache[6];
  //real otherWriteCache[3];
  __shared__ real L[3];
  __shared__ real cachedPos[3*numThreadsPerBlock];
  __shared__ real choleskyCache[12*numThreadsPerBlock];
  //const int lda=(((N*3)+31)/32)*32;
  //__shared__ real myresistance[6*numThreadsPerBlock];
  //real myresistance[6];
  //__shared__ real otherresistance[6*numThreadsPerBlock];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (threadIdx.x < 3){ // copy L to shared memory
    L[threadIdx.x]=L_g[threadIdx.x];
  }
  for (int l=0;l<3;l++){
    cachedPos[threadIdx.x+l*numThreadsPerBlock] = r[threadIdx.x+l*numThreadsPerBlock+blockIdx.x*blockDim.x*3];
    writeCache[l]= 0;
  }
  __syncthreads();
  for (int d=0;d<3;d++){
    mypos[d] = cachedPos[threadIdx.x*3+d];
  }
  
  for (int offset=0;offset<N;offset+=numThreadsPerBlock){
    // copy positions to shared memory
#pragma unroll
    for (int l=0;l<3;l++){
      cachedPos[threadIdx.x+l*numThreadsPerBlock] = r[threadIdx.x+l*numThreadsPerBlock+offset*3];
    }
    __syncthreads();
    for (int j=offset;j<min(offset+numThreadsPerBlock,N);j++){
      real dr[DIM];
      real dr2=0;
#pragma unroll
      for (int k=0;k<DIM;k++){
	dr[k]=mypos[k]-cachedPos[3*(j-offset)+k]; // r_ij
	dr[k]-=L[k]*rint(dr[k]/L[k]); // fold back
	dr2+=dr[k]*dr[k];
      }
#ifndef SD_RESISTANCE_CORRECT
#warning "SD Brownian motion only support corrected resistance calculation ..."
#endif
      real r2bcorr_diag_self     = 0;
      real r2bcorr_diag_mix      = 0;
      real r2bcorr_offdiag_self  = 0;
      real r2bcorr_offdiag_mix   = 0;

      int wasInLoop = 0;
      if (i >= N || i >= j || j >= N){
	writeCache[3]=0;
	writeCache[4]=0;
	writeCache[5]=0;
      }
      // j > i
      else if (dr2 < 4*a*4*a  && 2*a*2*a < dr2 ){// 2*a < drn < 4*a 
	wasInLoop = 1;
	// python code:
	// # Use only singular therms, namely to order O(s_ij^0)                                                                  
	// T=(1./4./s-1/4-9./40.*ls)*dr*drt/dr2
	// #           ^ this additonal constant is so that the mobility is smooth
	// # c.f. N.-Q. Nguyen and A. J. C. Ladd, PHYSICAL REVIEW E 66, 046708 (2002) equation (34)                               
	// T+=1./6.*ls*(-one+dr*drt/dr2)
	// R[3*i:3*i+3,3*j:3*j+3]=-T
	// R[3*i:3*i+3,3*i:3*i+3]+=T
	real drn= sqrt(dr2); // length of dr
	real s = drn/a-2;
	real ls = log(s);
	
	real const t_c=-0.125+9./40.*log(2.)+3./112.*2.*log(2.);
	real offdiag_fac =(-0.25/s+9./40.*ls+3./112.*s*ls-t_c)/dr2;
	real diag_fac    =(1./6.*ls);
	
	real dr4=dr2*dr2;
	real dr6=dr4*dr2;
	// constants for correction
	const real dr_c1 = 4;
	const real dr_c2 = 4*4;
	const real dr_c3 = 4*4*4;
	const real dr_c4 = 4*4*4*4;
	const real dr_c5 = 4*4*4*4*4;
	const real dr_c6 = 4*4*4*4*4*4;
	const real r2bcorr_diag_self_c    = (4.*dr_c6)/(4.*dr_c6-9.*dr_c4+12.*dr_c2-4.)         ;
	const real r2bcorr_diag_mix_c     = (9.*dr_c5-4.*dr_c3)/(4.*dr_c6-9.*dr_c4+12.*dr_c2-4.);
	const real r2bcorr_offdiag_self_c = 16.*dr_c2 /(16.*dr_c2-25)                            - 2./6.*log(2.);
	const real r2bcorr_offdiag_mix_c  = 20.*dr_c1 /(16.*dr_c2-25)                            - 2./6.*log(2.);
	// real computation
	r2bcorr_diag_self     = diag_fac    - 1./(1-9./4./dr2+3./dr4-1./dr6)                     + r2bcorr_diag_self_c;
	r2bcorr_diag_mix      = diag_fac    - (6.*dr4*drn-4.*dr2*drn)/(4.*dr6-9.*dr4+12.*dr2-4.) + r2bcorr_diag_mix_c;
	r2bcorr_offdiag_self  = offdiag_fac - 1./(1.-25./16./dr2)                                + r2bcorr_offdiag_self_c;
	r2bcorr_offdiag_mix   = offdiag_fac - 1./(16./20.*drn-25./20./drn)                       + r2bcorr_offdiag_mix_c;
	r2bcorr_diag_self    /= self_mobility;
	r2bcorr_diag_mix     /= self_mobility;
	r2bcorr_offdiag_self /= self_mobility;
	r2bcorr_offdiag_mix  /= self_mobility;
	
	// This is the cholesky decomposition.
	// note that we try to avoid the usage of registers, so we use shared mem
	// myCC is a makro, defined here to shorten the lines:
#define myCC(pos) choleskyCache[threadIdx.x+ (pos)*numThreadsPerBlock]
	// without it would look more like this:
	//choleskyCache[threadIdx.x+ 0*numThreadsPerBlock] = sqrt(r2bcorr_diag_self+r2bcorr_offdiag_self*dr[0]*dr[0]);
	//choleskyCache[threadIdx.x+ 1*numThreadsPerBlock] = r2bcorr_offdiag_self*dr[0]*dr[1] / choleskyCache[threadIdx.x+ 0*numThreadsPerBlock];
	// L_{1,1} to L_{6,1}
	myCC(0)  = sqrt(r2bcorr_diag_self+r2bcorr_offdiag_self*dr[0]*dr[0]);
	myCC(1)  =                        r2bcorr_offdiag_self*dr[0]*dr[1] / myCC(0);
	myCC(2)  =                        r2bcorr_offdiag_self*dr[0]*dr[2] / myCC(0);
	myCC(3)  =    (r2bcorr_diag_mix + r2bcorr_offdiag_mix *dr[0]*dr[0])/ myCC(0);
	myCC(4)  =                        r2bcorr_offdiag_mix *dr[0]*dr[1] / myCC(0);
	myCC(5)  =                        r2bcorr_offdiag_mix *dr[0]*dr[2] / myCC(0);
	
	writeCache[0]+=myCC(0)  * gaussian[0*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[1]+=myCC(1)  * gaussian[0*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[2]+=myCC(2)  * gaussian[0*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[3] =myCC(3)  * gaussian[0*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[4] =myCC(4)  * gaussian[0*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[5] =myCC(5)  * gaussian[0*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	// used: 6
	// L_{2,2} to L_{6,2}
	myCC(0)  = sqrt(r2bcorr_diag_self+r2bcorr_offdiag_self*dr[1]*dr[1] - SQR(    myCC(1)));
	myCC(6)  =                       (r2bcorr_offdiag_self*dr[1]*dr[2] - myCC(2)*myCC(1))/myCC(0);
	myCC(7)  =                       (r2bcorr_offdiag_mix *dr[1]*dr[0] - myCC(3)*myCC(1))/myCC(0);
	myCC(8)  =     (r2bcorr_diag_mix +r2bcorr_offdiag_mix *dr[1]*dr[1] - myCC(4)*myCC(1))/myCC(0);
	myCC(9)  =                       (r2bcorr_offdiag_mix *dr[1]*dr[2] - myCC(5)*myCC(1))/myCC(0);
	writeCache[1]+=myCC(0)  * gaussian[1*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[2]+=myCC(6)  * gaussian[1*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[3]+=myCC(7)  * gaussian[1*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[4]+=myCC(8)  * gaussian[1*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[5]+=myCC(9)  * gaussian[1*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	// used: 11 - 1
	// L_{3,3} to L_{6,3}
	myCC(0)  = sqrt(r2bcorr_diag_self+r2bcorr_offdiag_self*dr[2]*dr[2] - SQR(    myCC(2))- SQR(    myCC(6)));
	myCC(1)  =                       (r2bcorr_offdiag_mix *dr[2]*dr[0] - myCC(3)*myCC(2) - myCC(7)*myCC(6))/myCC(0);
	myCC(10) =                       (r2bcorr_offdiag_mix *dr[2]*dr[1] - myCC(4)*myCC(2) - myCC(8)*myCC(6))/myCC(0);
	myCC(11) =     (r2bcorr_diag_mix +r2bcorr_offdiag_mix *dr[2]*dr[2] - myCC(5)*myCC(2) - myCC(9)*myCC(6))/myCC(0);
	writeCache[2]+=myCC(0)  * gaussian[2*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[3]+=myCC(1)  * gaussian[2*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[4]+=myCC(10) * gaussian[2*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[5]+=myCC(11) * gaussian[2*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	// used: 15 - 3
	// L_{4,4} to L_{6,4}
	myCC(0)  = sqrt(r2bcorr_diag_mix +r2bcorr_offdiag_mix *dr[0]*dr[0] - SQR(    myCC(3))- SQR(    myCC(7))
			- SQR(     myCC(1)));
	myCC(2)  =                       (r2bcorr_offdiag_mix *dr[0]*dr[1] - myCC(4)*myCC(3) - myCC(8)*myCC(7) 
					  - myCC(10)*myCC(1))/myCC(0);
	myCC(6)  =                       (r2bcorr_offdiag_mix *dr[0]*dr[2] - myCC(5)*myCC(3) - myCC(9)*myCC(7) 
					  - myCC(11)*myCC(1))/myCC(0);
	writeCache[3]+=myCC(0)  * gaussian[3*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[4]+=myCC(2)  * gaussian[3*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[5]+=myCC(6)  * gaussian[3*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	// used: 18 - 6
	// L_{5,5} and L_{6,5}
	myCC(0)  = sqrt(r2bcorr_diag_mix +r2bcorr_offdiag_mix *dr[1]*dr[1] - SQR(    myCC(4))- SQR(    myCC(8))
			- SQR(     myCC(10))- SQR(    myCC(2)));
	myCC(3)  =                       (r2bcorr_offdiag_mix *dr[1]*dr[2] - myCC(5)*myCC(4) - myCC(9)*myCC(8) 
					  - myCC(11)*myCC(10) - myCC(6)*myCC(2))/myCC(0);
	writeCache[4]+=myCC(0)  * gaussian[4*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	writeCache[5]+=myCC(3)  * gaussian[4*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	// used: 20 - 10
	// L_{6,6} would be:
	myCC(0) = sqrt(r2bcorr_diag_mix +r2bcorr_offdiag_mix *dr[2]*dr[2] - SQR(myCC(5))    - SQR(myCC(9))     
		       - SQR(myCC(11)) - SQR(myCC(6)) - SQR(myCC(3)));
	writeCache[5]+=myCC(0)  * gaussian[5*gaussian_ldd+threadIdx.x+blockDim.x*blockIdx.x+6*gaussian_ldd*interactions];
	// used 21 - 15
	interactions++;
      }
      // for the particle j (writeCache[3-5]) we can reduce localy:
      
      int * haveInteraction = (int *) choleskyCache+6*numThreadsPerBlock; // reuse shared memory
      choleskyCache[threadIdx.x+0*numThreadsPerBlock]=writeCache[3];
      choleskyCache[threadIdx.x+1*numThreadsPerBlock]=0;
      choleskyCache[threadIdx.x+2*numThreadsPerBlock]=writeCache[4];
      choleskyCache[threadIdx.x+3*numThreadsPerBlock]=0;
      choleskyCache[threadIdx.x+4*numThreadsPerBlock]=writeCache[5];
      choleskyCache[threadIdx.x+5*numThreadsPerBlock]=0;
      haveInteraction[threadIdx.x]=wasInLoop;
      haveInteraction[threadIdx.x+numThreadsPerBlock]=0;
      for (int t=(blockDim.x+1)/2;t>1;t=(t+1)/2){
	if (threadIdx.x < t){
	  choleskyCache[threadIdx.x]+=choleskyCache[threadIdx.x+t];
	  choleskyCache[threadIdx.x+2*numThreadsPerBlock]+=choleskyCache[threadIdx.x+t +2*numThreadsPerBlock];
	  choleskyCache[threadIdx.x+4*numThreadsPerBlock]+=choleskyCache[threadIdx.x+t +2*numThreadsPerBlock];
	  haveInteraction[threadIdx.x]|=haveInteraction[threadIdx.x+t];
	  choleskyCache[threadIdx.x+t]=0;
	  choleskyCache[threadIdx.x+t +2*numThreadsPerBlock]=0;
	  choleskyCache[threadIdx.x+t +4*numThreadsPerBlock]=0;
	  haveInteraction[threadIdx.x+t]=0;
	}
	__syncthreads();
      }
      if (threadIdx.x==0){
	if (haveInteraction[0] || haveInteraction[1]){
	  choleskyCache[0]+=choleskyCache[1];
	  choleskyCache[2*numThreadsPerBlock]+=choleskyCache[1+2*numThreadsPerBlock];
	  choleskyCache[4*numThreadsPerBlock]+=choleskyCache[1+4*numThreadsPerBlock];
	  atomicAdd(brownian_force_nf+j*3,   choleskyCache[0]);
	  atomicAdd(brownian_force_nf+j*3+1, choleskyCache[2*numThreadsPerBlock]);
	  atomicAdd(brownian_force_nf+j*3+2, choleskyCache[4*numThreadsPerBlock]);
	}
      }
    }
  }
  if ( i < N){
#pragma unroll 3
    for (int k=0;k<3;k++){
      atomicAdd(brownian_force_nf+i*3+k, writeCache[k]);
    }
  }
}
// this adds the identity matrix to a given matrix of ld=size
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
// block : (ignored) the number of elements to process per thread
//         if this is e.g. 3 and the matrix is 3Nx3N, than N threads have to be started
__global__ void sd_add_identity_matrix(real * matrix, int size, int lda){
  //int lda=((size+31)/32)*32;
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  //for (int i = idx*block; i< (idx+1)*block; i++){
  for (int i = idx;i< size; i+=blockDim.x*gridDim.x){
    matrix[i+i*lda]+=1;
  }
}

// this sets a block to zero
// matrix: pointer to the given matrix
// size  : the size of the matrix (in the example below 3N)
__global__ void sd_set_zero_matrix(real * matrix, int size){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  int matsize=((size+31)/32)*32;
  matsize*=size;
  for (int i = idx;i< matsize; i+=blockDim.x*gridDim.x){
    matrix[i]=0;
  }
}


// this sets a block to zero
// data  : pointer to the given data
// size  : the size of the data
__global__ void sd_set_zero(real * data, int size){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  for (int i = idx;i< size; i+=blockDim.x*gridDim.x){
    data[i]=0;
  }
}

// this sets a block to zero
// data  : pointer to the given data
// size  : the size of the data
// value : the value written to the data block
__global__ void sd_set_int(int * data, int size, int value){
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  for (int i = idx;i< size; i+=blockDim.x*gridDim.x){
    data[i]=value;
  }
}




#define DIST (2+1e-1)
#define DISP_MAX (10000)

__global__ void sd_real_integrate_prepare( real * r_d , real * disp_d, real * L, real a, int N){
  /*for (int idx = blockIdx.x*blockDim.x + threadIdx.x;
       idx<N ;
       idx+=blockDim.x*gridDim.x){
    real disp2=0;
#pragma unroll
    for (int d=0;d<DIM;d++){
      disp2+=disp_d[idx*DIM+d]*disp_d[idx*DIM+d];
    }
    if (disp2 > DISP_MAX*DISP_MAX){
      real fac=DISP_MAX/sqrt(disp2);
#pragma unroll
      for (int d=0;d<DIM;d++){
	disp_d[idx*DIM+d]*=fac;
      }
    }
  }*/
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  i*=3;
  real disp2;
#pragma unroll
  for (int d=0;d<3;d++){
    disp2+=disp_d[i+d]*disp_d[i+d];
  }
  if (disp2> DISP_MAX*DISP_MAX){
    disp2=DISP_MAX/sqrt(disp2);
#pragma unroll
    for (int d=0;d<3;d++){
      disp_d[i+d]*=disp2;
    }
  }
}
__global__ void sd_real_integrate( real * r_d , real * disp_d, real * L, real a, int N)
{
  
  //for (int idx = blockIdx.x*blockDim.x + threadIdx.x;
  //     idx<N ;
  //     idx+=blockDim.x*gridDim.x){
  int idx =  blockIdx.x*blockDim.x + threadIdx.x;
  // t is the factor how far of disp_d we will move.
  // in case everything is fine, we will move t, if there is some trouble,
  // we will move less to avoid collision
  real t=1;
  real rnew[DIM];
  for (int d=0;d<DIM;d++){
    rnew[d]=r_d[DIM*idx+d]+disp_d[DIM*idx+d];
  }
  const real distmin=(3*a)*(3*a);
  for (int i=0;i<N;i++){
    if (idx==i){
      i++;
      if (i >N){
	continue;
      }
    }
    real dr2=0;
    for (int d=0;d<DIM;d++){
      real tmp=r_d[i*DIM+d]-rnew[d];
      tmp-=L[d]*rint(tmp/L[d]);
      dr2+=tmp*tmp;
    }
    if (dr2 <distmin){ // possible colision - check better
      dr2=0;
      //real dr2o=0; // or do we need old distance?
      for (int d=0;d<DIM;d++){
	real tmp=r_d[i*DIM+d]+disp_d[i*DIM+d]-rnew[d];
	tmp-=L[d]*rint(tmp/L[d]);
	dr2+=tmp*tmp;
	//tmp=r_d[i*DIM+d]-r_d[idx*DIM+d];
	//tmp-=L*rint(tmp/L);
	//dr2o+=tmp*tmp;
      }
      if (dr2 < DIST*DIST*a*a){ // do they collide after the step?
	// ideal: the motion which is responsible for the crash: avoid it.
	// just move them that far that they nearly touch each other.
	// therefore we need the soluten of an quadratic equation
	// in case they are already closer than DIST*a this will move them appart.
	// first: get the coefficents
	real alpha=0,beta=0,gamma=0;
	for (int d=0;d<DIM;d++){
	  real t1=r_d[i*DIM+d]-r_d[idx*DIM+d];
	  t1-=L[d]*rint(t1/L[d]);
	  real t2=disp_d[i*DIM+d]-disp_d[idx*DIM+d];
	  //t2-=L*rint(t2/L); // we would have a problem if we would need to fold back these ...
	  alpha +=t2*t2;
	  beta  +=2*t1*t2;
	  gamma +=t1*t1;
	} 
	// now we want to solve for t: alpha*t**2+beta*t+gamma=DIST*a
	// we want the solution with the minus in the 'mitternachtsformel'
	// because the other solution is when the particles moved through each other
	real tnew = (-beta-sqrt(beta*beta-4*alpha*gamma))/(2*alpha);
	if (tnew < t){ // use the smallest t
	  t=tnew;
	}
      }
    }
  }
  for (int d=0;d<DIM;d++){ // actually do the integration
    r_d[DIM*idx+d]+=disp_d[DIM*idx+d]*t;
  }
  //#warning "Debug is still enabaled"
    //pos_d[DIM*N+idx]=t;
}

__global__ void sd_bucket_sort( real * pos , real * bucketSize, int * bucketNum, int N,
				int * particleCount, int * particleList, int maxParticlePerCell, int totalBucketNum){
  for (int i = blockIdx.x*blockDim.x + threadIdx.x;
       i<N ;
       i+=blockDim.x*gridDim.x){
    int3 bucket;
#pragma unroll 3
    for (int d =0; d<3; d++){
      real tmp;
      // no asm version:
      // tmp = pos[i*3+d];
      // asm version avoids caching
#ifdef SD_USE_FLOAT
      asm("ld.global.cs.f32 %0,[%1];\n"
	: "=f"(tmp) : "l"(pos+i*3+d) : );
#else
      asm("ld.global.cs.f64 %0,[%1];\n"
	: "=d"(tmp) : "l"(pos+i*3+d) : );
#endif
      tmp/=bucketSize[d];
      int x;
      // this should work - but somehow it does not compile
      x=__real2int_rd(tmp);
      // the following code is an replacement ...
      // but with this the loop is not getting unrolled
      //asm("cvt.rmi.s32.f64 %0, %1;\n"
      //    : "=r"(x) : "d"(tmp) : );
      // this should also work.
      // but the corresponding ptx code first rounds, and then converts in a second step ...
      // this could lead to rounding errors ...
      //x=floor(tmp);
      //x%=bucketNum[d];
      // avoid negativ numbers
      x= (x < 0)?x+bucketNum[d]: x;
      //x+=bucketNum[d];
      //x%=bucketNum[d];
      switch (d){
      case 0:
	bucket.x = x;
	break;
      case 1:
	bucket.y = x;
	break;
      case 2:
	bucket.z = x;
	break;
      }
    }
    int myBucket = bucket.x + bucket.y*bucketNum[0] + bucket.z*bucketNum[0]*bucketNum[1];
    int num = atomicAdd(particleList+myBucket, 1);
    if (num < maxParticlePerCell){ // every thread should do this - so this is not a branch ...
      particleList[myBucket+num*totalBucketNum]=i;
    }else{
      // Note: printf in device code works only with cc>=2.0 //
#if (__CUDA_ARCH__>=200)
      printf("error: overflow in grid cell (%i,%i,%i)\n",bucket.x,bucket.y,bucket.z);
#endif
    }
  }
}


/* *************************************************************************************************************** *
 * ********************************************    DEVICE-Functions   ******************************************** *
 * *************************************************************************************************************** */

__device__ double atomicAdd(double * address, double inc){

ull *addressUll = (ull*) address;
ull oldValue=*addressUll;
ull assumedValue;
do {
assumedValue=oldValue;
ull newValue = __double_as_longlong (__longlong_as_double(assumedValue)+inc);
oldValue = atomicCAS(addressUll,assumedValue,newValue);
}
  while (oldValue != assumedValue);
return __longlong_as_double(oldValue);
}

#endif
